#include "hip/hip_runtime.h"
#include "caffe2/core/THCCachingAllocator_gpu.h"

#include <deque>
#include <map>
#include <memory>
#include <mutex>
#include <set>
#include <unordered_map>

#include <hip/hip_runtime_api.h>

#include "caffe2/core/context_gpu.h"

//
// Yet another caching allocator for CUDA device allocations.
//
// - The allocator attempts to find the smallest cached block that will fit the
//   requested size. If the block is larger than the requested size, it may be
//   split. If no block is found, the allocator will delegate to hipMalloc.
// - If the hipMalloc fails, the allocator will free all cached blocks that
//   are not split and retry the allocation.
// - Large (>1MB) and small allocation requests are handled separately. Large
//   allocation requests can be filled by a hipMalloc call of the exact size.
//   Small requests will allocate and split a 1MB buffer, if necessary.
//
// With this allocator, allocations and frees should logically be considered
// "usages" of the memory segment associated with streams, just like kernel
// launches. The programmer must insert the proper synchronization if memory
// segments are used from multiple streams.
//
// Thread Safety: the allocator is NOT thread safe. Calls to { Alloc, Free }
// must be synchronized by the programmer.
//

namespace {

const size_t kRoundSmall = 512; // round up small allocs to 512 bytes
const size_t kRoundLarge = 131072; // round up large allocs to 128 KiB
const size_t kSmallAlloc = 1048576; // largest "small" allocation is 1 MiB

struct Block {
  int device; // gpu
  hipStream_t stream; // allocation stream
  size_t size; // block size in bytes
  char* ptr; // memory address
  bool allocated; // in-use flag
  Block* prev; // prev block if split from a larger allocation
  Block* next; // next block if split from a larger allocation
  int event_count; // number of outstanding CUDA events

  Block(int device, hipStream_t stream, size_t size, char* ptr = nullptr)
      : device(device),
        stream(stream),
        size(size),
        ptr(ptr),
        allocated(0),
        prev(nullptr),
        next(nullptr),
        event_count(0) {}
};

static bool BlockComparator(const Block* a, const Block* b) {
  if (a->device != b->device) {
    return a->device < b->device;
  }
  if (a->stream != b->stream) {
    return (uintptr_t)a->stream < (uintptr_t)b->stream;
  }
  if (a->size != b->size) {
    return a->size < b->size;
  }
  return (uintptr_t)a->ptr < (uintptr_t)b->ptr;
}

static size_t roundSize(size_t size) {
  if (size < kRoundSmall) {
    size = kRoundSmall;
  } else if (size < kSmallAlloc) {
    size += kRoundSmall - 1 - (size - 1) % kRoundSmall;
  } else {
    size += kRoundLarge - 1 - (size - 1) % kRoundLarge;
  }
  return size;
}

} // namespace

namespace caffe2 {

struct THCCachingAllocatorImpl {
  typedef bool (*Comparison)(const Block*, const Block*);
  typedef std::set<Block*, Comparison> FreeBlocks;

  // lock around all operations
  std::mutex mutex;

  // cached blocks larger than 1 MB
  FreeBlocks largeBlocks_;

  // cached blocks 1 MB or smaller
  FreeBlocks smallBlocks_;

  // allocated blocks by device pointer
  std::unordered_map<void*, Block*> allocatedBlocks_;

  THCCachingAllocatorImpl()
      : largeBlocks_(BlockComparator), smallBlocks_(BlockComparator) {}

  ~THCCachingAllocatorImpl() {
    emptyCache();
  }

  /** allocates a block which is safe to use from the provided stream */
  hipError_t Alloc(void** devPtr, size_t size, hipStream_t stream) {
    int device;
    hipError_t err = hipGetDevice(&device);
    if (err != hipSuccess) {
      return err;
    }

    size = roundSize(size);
    bool small = size <= kSmallAlloc;

    Block search_key(device, stream, size);
    auto& free_blocks = small ? smallBlocks_ : largeBlocks_;

    Block* block = nullptr;
    Block* remaining = nullptr;

    auto it = free_blocks.lower_bound(&search_key);
    if (it != free_blocks.end() && (*it)->device == device &&
        (*it)->stream == stream) {
      block = *it;
      free_blocks.erase(it);
    } else {
      void* ptr;
      size_t alloc_size = small ? kSmallAlloc : size;
      err = cudaMallocRetry(device, &ptr, alloc_size);
      if (err != hipSuccess) {
        return err;
      }
      block = new Block(device, stream, alloc_size, (char*)ptr);
    }

    if (block->size - size >= (small ? kRoundSmall : kSmallAlloc + 1)) {
      remaining = block;

      block = new Block(device, stream, size, block->ptr);
      block->prev = remaining->prev;
      if (block->prev) {
        block->prev->next = block;
      }
      block->next = remaining;

      remaining->prev = block;
      remaining->ptr += size;
      remaining->size -= size;
      free_blocks.insert(remaining);
    }

    block->allocated = true;
    allocatedBlocks_[block->ptr] = block;

    *devPtr = (void*)block->ptr;
    return hipSuccess;
  }

  hipError_t Free(void* ptr) {
    if (!ptr) {
      return hipSuccess;
    }

    auto it = allocatedBlocks_.find(ptr);
    if (it == allocatedBlocks_.end()) {
      return hipErrorInvalidDevicePointer;
    }

    Block* block = it->second;
    allocatedBlocks_.erase(it);
    block->allocated = false;

    freeBlock(block);
    return hipSuccess;
  }

  /** returns cached blocks to the system allocator */
  hipError_t emptyCache() {
    hipError_t err =
        freeBlocks(largeBlocks_, largeBlocks_.begin(), largeBlocks_.end());
    if (err != hipSuccess) {
      return err;
    }
    err = freeBlocks(smallBlocks_, smallBlocks_.begin(), smallBlocks_.end());
    if (err != hipSuccess) {
      return err;
    }
    return hipSuccess;
  }

  /** moves a block into the free block list */
  void freeBlock(Block* block) {
    CAFFE_ENFORCE(!block->allocated && block->event_count == 0);
    bool small = block->size <= kSmallAlloc;
    auto& free_blocks = small ? smallBlocks_ : largeBlocks_;
    tryMergeBlocks(block, block->prev, free_blocks);
    tryMergeBlocks(block, block->next, free_blocks);
    free_blocks.insert(block);
  }

  /** combine previously split blocks */
  void tryMergeBlocks(Block* dst, Block* src, FreeBlocks& free_blocks) {
    if (!src || src->allocated || src->event_count > 0) {
      return;
    }
    if (dst->prev == src) {
      dst->ptr = src->ptr;
      dst->prev = src->prev;
      if (dst->prev) {
        dst->prev->next = dst;
      }
    } else {
      dst->next = src->next;
      if (dst->next) {
        dst->next->prev = dst;
      }
    }
    dst->size += src->size;
    free_blocks.erase(src);
    delete src;
  }

  hipError_t cudaMallocRetry(int device, void** devPtr, size_t size) {
    // Try hipMalloc. If hipMalloc fails, frees all non-split cached blocks
    // and retries.
    hipError_t err = hipMalloc(devPtr, size);
    if (err != hipSuccess) {
      hipGetLastError();
      err = freeCachedBlocks(device);
      if (err != hipSuccess) {
        return err;
      }
      err = hipMalloc(devPtr, size);
      if (err != hipSuccess) {
        return err;
      }
    }
    return hipSuccess;
  }

  hipError_t freeCachedBlocks(int device) {
    // Free all non-split cached blocks on device
    Block lower_bound(device, nullptr, 0);
    Block upper_bound(device + 1, nullptr, 0);

    hipError_t err = freeBlocks(
        largeBlocks_,
        largeBlocks_.lower_bound(&lower_bound),
        largeBlocks_.lower_bound(&upper_bound));
    if (err != hipSuccess) {
      return err;
    }
    err = freeBlocks(
        smallBlocks_,
        smallBlocks_.lower_bound(&lower_bound),
        smallBlocks_.lower_bound(&upper_bound));
    return err;
  }

  hipError_t freeBlocks(
      FreeBlocks& blocks,
      FreeBlocks::iterator it,
      FreeBlocks::iterator end) {
    // Frees all non-split blocks between `it` and `end`
    while (it != end) {
      Block* block = *it;
      if (!block->prev && !block->next) {
        hipError_t err = hipFree((void*)block->ptr);
        if (err != hipSuccess) {
          return err;
        }
        auto cur = it;
        ++it;
        blocks.erase(cur);
        delete block;
      } else {
        ++it;
      }
    }
    return hipSuccess;
  }
};

THCCachingAllocator::THCCachingAllocator()
    : _impl(new THCCachingAllocatorImpl()) {}

THCCachingAllocator::~THCCachingAllocator() {
  delete _impl;
}

hipError_t
THCCachingAllocator::Alloc(void** refPtr, size_t nbytes, hipStream_t stream) {
  return _impl->Alloc(refPtr, nbytes, stream);
}

hipError_t THCCachingAllocator::Free(void* ptr) {
  return _impl->Free(ptr);
}

} // namespace caffe2
