#include "hip/hip_runtime.h"
#include "caffe2/operators/layer_norm_op.h"

#include <hipcub/hipcub.hpp>

#include "caffe2/core/context_gpu.h"
#include "caffe2/utils/math.h"
#include "caffe2/utils/math_utils.h"

namespace caffe2 {

namespace {

template <typename T>
using BlockReduce = hipcub::BlockReduce<T, CAFFE_CUDA_NUM_THREADS>;

template <typename T>
__global__ void ComputeStdDevAndFusedParamsCUDAKernel(
    const int N,
    const T epsilon,
    const T* mean,
    const T* var,
    T* stddev,
    T* scale,
    T* bias);

template <>
__global__ void ComputeStdDevAndFusedParamsCUDAKernel<float>(
    const int N,
    const float epsilon,
    const float* mean,
    const float* var,
    float* stddev,
    float* scale,
    float* bias) {
  CUDA_1D_KERNEL_LOOP(i, N) {
#if __CUDA_ARCH__ >= 350
    const float rstd = rsqrtf(__ldg(var + i) + epsilon);
    stddev[i] = rstd * (__ldg(var + i) + epsilon);
    scale[i] = rstd;
    bias[i] = -rstd * __ldg(mean + i);
#else
    const float rstd = rsqrtf(var[i] + epsilon);
    stddev[i] = rstd * (var[i] + epsilon);
    scale[i] = rstd;
    bias[i] = -rstd * mean[i];
#endif
  }
}

template <typename T>
__global__ void LayerNormForwardCUDAKernel(
    const int M,
    const int N,
    const T* X,
    const T* scale,
    const T* bias,
    T* Y) {
  for (int i = blockIdx.x; i < M; i += gridDim.x) {
#if __CUDA_ARCH__ >= 350
    const float scale_val = __ldg(scale + i);
    const float bias_val = __ldg(bias + i);
#else
    const float scale_val = scale[i];
    const float bias_val = bias[i];
#endif
    for (int j = threadIdx.x; j < N; j += blockDim.x) {
      const int index = i * N + j;
#if __CUDA_ARCH__ >= 350
      Y[index] = __ldg(X + index) * scale_val + bias_val;
#else
      Y[index] = X[index] * scale_val + bias_val;
#endif
    }
  }
}

template <typename T>
__global__ void ComputeInternalGradientsCUDAKernel(
    const int M,
    const int N,
    const T* dY,
    const T* X,
    T* ds,
    T* db) {
  __shared__ typename BlockReduce<T>::TempStorage ds_storage;
  __shared__ typename BlockReduce<T>::TempStorage db_storage;
  for (int i = blockIdx.x; i < M; i += gridDim.x) {
    T ds_val = 0;
    T db_val = 0;
    for (int j = threadIdx.x; j < N; j += blockDim.x) {
      const int index = i * N + j;
#if __CUDA_ARCH__ >= 350
      ds_val += __ldg(dY + index) * __ldg(X + index);
      db_val += __ldg(dY + index);
#else
      ds_val += dY[index] * X[index];
      db_val += dY[index];
#endif
    }
    ds_val = BlockReduce<T>(ds_storage).Sum(ds_val);
    db_val = BlockReduce<T>(db_storage).Sum(db_val);
    if (threadIdx.x == 0) {
      ds[i] = ds_val;
      db[i] = db_val;
    }
    __syncthreads();
  }
}

template <typename T>
__global__ void ComputeFusedParamsCUDAKernel(
    const int M,
    const int N,
    const T* mean,
    const T* sig,
    const T* ds,
    const T* db,
    T* dY_scale,
    T* X_scale,
    T* bias) {
  const T scale = T(1) / static_cast<T>(N);
  CUDA_1D_KERNEL_LOOP(i, M) {
#if __CUDA_ARCH__ >= 350
    const T rsig = T(1) / __ldg(sig + i);
    const T X_scale_val = (__ldg(db + i) * __ldg(mean + i) - __ldg(ds + i)) *
        math::utils::Cube<T>(rsig) * scale;
    dY_scale[i] = rsig;
    X_scale[i] = X_scale_val;
    bias[i] = -X_scale_val * __ldg(mean + i) - __ldg(db + i) * rsig * scale;
#else
    const T rsig = T(1) / sig[i];
    const T X_scale_val =
        (db[i] * mean[i] - ds[i]) * math::utils::Cube<T>(rsig) * scale;
    dY_scale[i] = rsig;
    X_scale[i] = X_scale_val;
    bias[i] = -X_scale_val * mean[i] - db[i] * rsig * scale;
#endif
  }
}

template <typename T>
__global__ void LayerNormBackwardCUDAKenrel(
    const int M,
    const int N,
    const T* dY_scale,
    const T* dY,
    const T* X_scale,
    const T* X,
    const T* bias,
    T* dX) {
  for (int i = blockIdx.x; i < M; i += gridDim.x) {
#if __CUDA_ARCH__ >= 350
    const float dY_scale_val = __ldg(dY_scale + i);
    const float X_scale_val = __ldg(X_scale + i);
    const float bias_val = __ldg(bias + i);
#else
    const float dY_scale_val = dY_scale[i];
    const float X_scale_val = X_scale[i];
    const float bias_val = bias[i];
#endif
    for (int j = threadIdx.x; j < N; j += blockDim.x) {
      const int index = i * N + j;
#if __CUDA_ARCH__ >= 350
      dX[index] = __ldg(dY + index) * dY_scale_val +
          __ldg(X + index) * X_scale_val + bias_val;
#else
      dX[index] = dY[index] * dY_scale_val + X[index] * X_scale_val + bias_val;
#endif
    }
  }
}

} //  namespace

template <>
template <typename T>
void LayerNormOp<HIPContext>::ComputeStdDevAndFusedParams(
    const int N,
    const T* mean,
    const T* var,
    T* stddev,
    T* scale,
    T* bias) {
  ComputeStdDevAndFusedParamsCUDAKernel<T>
      <<<CAFFE_GET_BLOCKS(N),
         CAFFE_CUDA_NUM_THREADS,
         0,
         context_.hip_stream()>>>(
          N, static_cast<T>(epsilon_), mean, var, stddev, scale, bias);
}

template <>
template <typename T>
void LayerNormOp<HIPContext>::LayerNormForward(
    const int M,
    const int N,
    const T* X,
    const T* scale,
    const T* bias,
    T* Y) {
  LayerNormForwardCUDAKernel<T>
      <<<std::min(M, CAFFE_MAXIMUM_NUM_BLOCKS),
         CAFFE_CUDA_NUM_THREADS,
         0,
         context_.hip_stream()>>>(M, N, X, scale, bias, Y);
}

REGISTER_HIP_OPERATOR(LayerNorm, LayerNormOp<HIPContext>);

template <>
template <typename T>
void LayerNormGradientOp<HIPContext>::ComputeInternalGradients(
    const int M,
    const int N,
    const T* dY,
    const T* X,
    T* ds,
    T* db) {
  ComputeInternalGradientsCUDAKernel<T>
      <<<std::min(M, CAFFE_MAXIMUM_NUM_BLOCKS),
         CAFFE_CUDA_NUM_THREADS,
         0,
         context_.hip_stream()>>>(M, N, dY, X, ds, db);
}

template <>
template <typename T>
void LayerNormGradientOp<HIPContext>::ComputeFusedParams(
    const int M,
    const int N,
    const T* mean,
    const T* sig,
    const T* ds,
    const T* db,
    T* dY_scale,
    T* X_scale,
    T* bias) {
  ComputeFusedParamsCUDAKernel<T>
      <<<CAFFE_GET_BLOCKS(M),
         CAFFE_CUDA_NUM_THREADS,
         0,
         context_.hip_stream()>>>(
          M, N, mean, sig, ds, db, dY_scale, X_scale, bias);
}

template <>
template <typename T>
void LayerNormGradientOp<HIPContext>::LayerNormBackward(
    const int M,
    const int N,
    const T* dY_scale,
    const T* dY,
    const T* X_scale,
    const T* X,
    const T* bias,
    T* dX) {
  LayerNormBackwardCUDAKenrel<T>
      <<<std::min(M, CAFFE_MAXIMUM_NUM_BLOCKS),
         CAFFE_CUDA_NUM_THREADS,
         0,
         context_.hip_stream()>>>(M, N, dY_scale, dY, X_scale, X, bias, dX);
}

REGISTER_HIP_OPERATOR(LayerNormGradient, LayerNormGradientOp<HIPContext>);

} // namespace caffe2
