#include "hip/hip_runtime.h"
#include "caffe2/core/context_gpu.h"
#include "caffe2/operators/lengths_tile_op.h"

namespace caffe2 {

template <typename T>
__global__ void lengthsTileKernel(
    int numElements,
    int rowSize,
    const T* input,
    T* output,
    const int32_t* inputRowOffsets) {
  CUDA_1D_KERNEL_LOOP(i, numElements) {
    auto outputRowIndex = i / rowSize;
    auto inputBlockOffset = inputRowOffsets[outputRowIndex];
    auto indexInRow = i - outputRowIndex * rowSize;
    output[i] = input[inputBlockOffset + indexInRow];
  }
}

template <>
bool LengthsTileOp<HIPContext>::RunOnDevice() {
  auto& data = Input(DATA);
  auto& lengths = Input(LENGTHS);
  auto* output = Output(0);

  CAFFE_ENFORCE_EQ(lengths.ndim(), 1, "LENGTHS must be 1-D");
  CAFFE_ENFORCE_GE(data.ndim(), 1, "DATA should be at least 1-D");
  CAFFE_ENFORCE_EQ(lengths.size(), data.dim(0));

  lengths_host_.CopyFrom(lengths); // sync copy
  auto lengths_size = lengths_host_.size();
  auto* lengths_data = lengths_host_.data<int32_t>();

  int32_t total_length = 0;
  CPUContext cpuContext;
  math::Sum<int32_t, CPUContext>(
      lengths_size, lengths_data, &total_length, &cpuContext);

  auto shape = data.dims().vec();
  shape[0] = total_length;
  output->Resize(shape);

  auto numElementsPerRow = data.size_from_dim(1);
  auto numElements = total_length * numElementsPerRow;
  auto numBlocks = CAFFE_GET_BLOCKS(numElements);

  rowMappingHost_.Resize(total_length);
  rowMappingDevice_.Resize(total_length);
  auto* rowOffsets = rowMappingHost_.mutable_data<int32_t>();
  int32_t outputRow = 0;
  for (int64_t i = 0; i < lengths_size; i++) {
    auto length = lengths_data[i];
    for (int32_t j = 0; j < length; j++) {
      rowOffsets[outputRow++] = i * numElementsPerRow;
    }
  }

  context_.CopyFromCPU<int32_t>(
      total_length,
      rowMappingHost_.data<int32_t>(),
      rowMappingDevice_.mutable_data<int32_t>());
  context_.FinishDeviceComputation();

  if (data.template IsType<float>()) {
    lengthsTileKernel<<<
        numBlocks,
        CAFFE_CUDA_NUM_THREADS,
        0,
        context_.hip_stream()>>>(
        numElements,
        numElementsPerRow,
        data.data<float>(),
        output->mutable_data<float>(),
        rowMappingDevice_.data<int32_t>());
  } else if (data.template IsType<int>()) {
    lengthsTileKernel<<<
        numBlocks,
        CAFFE_CUDA_NUM_THREADS,
        0,
        context_.hip_stream()>>>(
        numElements,
        numElementsPerRow,
        data.data<int>(),
        output->mutable_data<int>(),
        rowMappingDevice_.data<int32_t>());
  } else if (data.template IsType<int64_t>()) {
    lengthsTileKernel<<<
        numBlocks,
        CAFFE_CUDA_NUM_THREADS,
        0,
        context_.hip_stream()>>>(
        numElements,
        numElementsPerRow,
        data.data<int64_t>(),
        output->mutable_data<int64_t>(),
        rowMappingDevice_.data<int32_t>());
  } else {
    CAFFE_THROW(
        "LengthsTile operator only supports 32-bit float, int and int64_t"
        " types but input was of type ",
        data.meta().name());
  }
  return true;
}

REGISTER_HIP_OPERATOR(LengthsTile, LengthsTileOp<HIPContext>);

} // namespace caffe2
