#include "hip/hip_runtime.h"
// Implements the math functions for GPU.

#include "caffe2/utils/math.h"

#include <cstring>
#include <limits>
#include <numeric>
#include <vector>

#include <cub/block/block_reduce.cuh>
#include <hipcub/hipcub.hpp>

#include <thrust/device_vector.h>
#include <thrust/functional.h>

#include "caffe2/core/context_gpu.h"
#include "caffe2/utils/conversions.h"

#include "caffe2/utils/fixed_divisor.h"
// TODO: Move this to fixed_divisor.h
#ifdef __HIPCC__
#define FIXED_DIVISOR int32_t
#define FIXED_DIVISOR_DIV(d, n) (n / d)
#define FIXED_DIVISOR_MOD(d, n) (n % d)
#define FIXED_DIVISOR_DIV_MOD(d, n, q, r) \
  do {                                    \
    const auto n_copy = n;                \
    *q = n_copy / d;                      \
    *r = n_copy % d;                      \
  } while (0)
#else // __HIPCC__
#define FIXED_DIVISOR FixedDivisor<int32_t>
#define FIXED_DIVISOR_DIV(d, n) (d.Div(n))
#define FIXED_DIVISOR_MOD(d, n) (d.Mod(n))
#define FIXED_DIVISOR_DIV_MOD(d, n, q, r) (d.DivMod(n, q, r))
#endif // __HIPCC__

#include "caffe2/utils/math_utils.h"

#if THRUST_VERSION >= 100800
#define THRUST_SUPPORTS_PER_THREAD
#endif // THRUST_VERSION >= 100800

namespace caffe2 {
namespace math {

namespace {

#define DELEGATE_SIMPLE_HOST_DEVICE_BINARY_FUNCTOR(Func, expr)          \
  template <typename T>                                                 \
  struct Func##Functor {                                                \
    inline __host__ __device__ T                                        \
    operator()(const T& lhs, const T& rhs) const {                      \
      return lhs expr rhs;                                              \
    }                                                                   \
  };                                                                    \
  template <>                                                           \
  struct Func##Functor<at::Half> {                                      \
    inline __host__ __device__ at::Half operator()(                     \
        const at::Half& lhs,                                            \
        const at::Half& rhs) const {                                    \
      return convert::To<float, at::Half>(convert::To<at::Half, float>( \
          lhs) expr convert::To<at::Half, float>(rhs));                 \
    }                                                                   \
  };
DELEGATE_SIMPLE_HOST_DEVICE_BINARY_FUNCTOR(Add, +)
DELEGATE_SIMPLE_HOST_DEVICE_BINARY_FUNCTOR(Sub, -)
DELEGATE_SIMPLE_HOST_DEVICE_BINARY_FUNCTOR(Mul, *)
DELEGATE_SIMPLE_HOST_DEVICE_BINARY_FUNCTOR(Div, /)
#undef DELEGATE_SIMPLE_HOST_DEVICE_BINARY_FUNCTOR

template <typename T>
__global__ void SinCosCUDAKernel(const int N, const T* X, T* S, T* C) {
  CUDA_1D_KERNEL_LOOP(i, N) {
#if __CUDA_ARCH__ >= 350
    c10::cuda::compat::sincos(__ldg(X + i), S + i, C + i);
#else
    c10::cuda::compat::sincos(X[i], S + i, C + i);
#endif
  }
}

template <typename TIn, typename TOut, class BinaryOperator>
__global__ void SimpleBinaryOpCUDAKernel(
    const int N,
    const BinaryOperator op,
    const TIn* A,
    const TIn* B,
    TOut* C) {
  CUDA_1D_KERNEL_LOOP(i, N) {
    C[i] = op(A[i], B[i]);
  }
}

template <typename TIn, typename TOut, class BinaryOperator, bool broadcast_1st>
__global__ void RowwiseBinaryOpCUDAKenel(
    const int size,
    const FIXED_DIVISOR cols,
    const BinaryOperator op,
    const TIn* A,
    const TIn* B,
    TOut* C) {
  CUDA_1D_KERNEL_LOOP(C_index, size) {
    const int j = FIXED_DIVISOR_MOD(cols, C_index);
    const int A_index = broadcast_1st ? j : C_index;
    const int B_index = broadcast_1st ? C_index : j;
    C[C_index] = op(A[A_index], B[B_index]);
  }
}

template <typename TIn, typename TOut, class BinaryOperator, bool broadcast_1st>
__global__ void ColwiseBinaryOpCUDAKenel(
    const int size,
    const FIXED_DIVISOR cols,
    const BinaryOperator op,
    const TIn* A,
    const TIn* B,
    TOut* C) {
  CUDA_1D_KERNEL_LOOP(C_index, size) {
    const int i = FIXED_DIVISOR_DIV(cols, C_index);
    const int A_index = broadcast_1st ? i : C_index;
    const int B_index = broadcast_1st ? C_index : i;
    C[C_index] = op(A[A_index], B[B_index]);
  }
}

template <typename TIn, typename TOut, class BinaryOperator, int D>
__global__ void BroadcastBinaryOpCUDAKernel(
    const int size,
    const SimpleArray<int, D> A_strides,
    const SimpleArray<int, D> B_strides,
    const SimpleArray<FIXED_DIVISOR, D> C_dims,
    const BinaryOperator op,
    const TIn* A,
    const TIn* B,
    TOut* C) {
  CUDA_1D_KERNEL_LOOP(C_index, size) {
    int A_index = 0;
    int B_index = 0;
    int C_index_val = C_index;
#pragma unroll
    for (int i = D - 1; i >= 0; --i) {
      int d;
      FIXED_DIVISOR_DIV_MOD(C_dims.data[i], C_index_val, &C_index_val, &d);
      A_index += d * A_strides.data[i];
      B_index += d * B_strides.data[i];
    }
    C[C_index] = op(A[A_index], B[B_index]);
  }
}

template <typename TIn, typename TOut, class BinaryOperator>
CAFFE2_CUDA_EXPORT void BinaryOpWith2DBroadcasting(
    const int rows,
    const int cols,
    const bool rowwise_broadcast,
    const bool broadcast_1st,
    const BinaryOperator& op,
    const TIn* A,
    const TIn* B,
    TOut* C,
    HIPContext* context) {
  if (rows == 0 || cols == 0) {
    return;
  }
  const int size = rows * cols;
  const FIXED_DIVISOR cols_div(cols);
  if (rowwise_broadcast) {
    if (broadcast_1st) {
      RowwiseBinaryOpCUDAKenel<TIn, TOut, BinaryOperator, true>
          <<<CAFFE_GET_BLOCKS(size),
             CAFFE_CUDA_NUM_THREADS,
             0,
             context->hip_stream()>>>(size, cols_div, op, A, B, C);
    } else {
      RowwiseBinaryOpCUDAKenel<TIn, TOut, BinaryOperator, false>
          <<<CAFFE_GET_BLOCKS(size),
             CAFFE_CUDA_NUM_THREADS,
             0,
             context->hip_stream()>>>(size, cols_div, op, A, B, C);
    }
  } else {
    if (broadcast_1st) {
      ColwiseBinaryOpCUDAKenel<TIn, TOut, BinaryOperator, true>
          <<<CAFFE_GET_BLOCKS(size),
             CAFFE_CUDA_NUM_THREADS,
             0,
             context->hip_stream()>>>(size, cols_div, op, A, B, C);
    } else {
      ColwiseBinaryOpCUDAKenel<TIn, TOut, BinaryOperator, false>
          <<<CAFFE_GET_BLOCKS(size),
             CAFFE_CUDA_NUM_THREADS,
             0,
             context->hip_stream()>>>(size, cols_div, op, A, B, C);
    }
  }
}

template <typename TIn, typename TOut, class BinaryOperator, int D>
CAFFE2_CUDA_EXPORT void BroadcastBinaryOpImpl(
    const int* A_dims,
    const int* B_dims,
    const int* C_dims,
    const BinaryOperator& op,
    const TIn* A,
    const TIn* B,
    TOut* C,
    HIPContext* context) {
  SimpleArray<int, D> A_strides_array;
  SimpleArray<int, D> B_strides_array;
  SimpleArray<FIXED_DIVISOR, D> C_dims_array;
  int A_stride = 1;
  int B_stride = 1;
  for (int i = D - 1; i >= 0; --i) {
    if (C_dims[i] == 0) {
      return;
    }
    A_strides_array.data[i] = A_dims[i] == 1 ? 0 : A_stride;
    B_strides_array.data[i] = B_dims[i] == 1 ? 0 : B_stride;
    A_stride *= A_dims[i];
    B_stride *= B_dims[i];
    C_dims_array.data[i] = FIXED_DIVISOR(C_dims[i]);
  }
  const int size =
      std::accumulate(C_dims, C_dims + D, 1, std::multiplies<int>());
  BroadcastBinaryOpCUDAKernel<TIn, TOut, BinaryOperator, D>
      <<<CAFFE_GET_BLOCKS(size),
         CAFFE_CUDA_NUM_THREADS,
         0,
         context->hip_stream()>>>(
          size, A_strides_array, B_strides_array, C_dims_array, op, A, B, C);
}

template <typename TIn, typename TOut, class BinaryOperator>
CAFFE2_CUDA_EXPORT void BroadcastBinaryOp(
    const int A_ndim,
    const int* A_dims,
    const int B_ndim,
    const int* B_dims,
    const BinaryOperator& op,
    const TIn* A,
    const TIn* B,
    TOut* C,
    HIPContext* context) {
  const int ndim = std::max(A_ndim, B_ndim);
  std::vector<int> A_dims_array(ndim);
  std::vector<int> B_dims_array(ndim);
  std::vector<int> C_dims_array(ndim);
  utils::ComputeBroadcastBinaryOpDims(
      A_ndim,
      A_dims,
      B_ndim,
      B_dims,
      A_dims_array.data(),
      B_dims_array.data(),
      C_dims_array.data());
  if (A_dims_array == B_dims_array) {
    const int size = std::accumulate(
        C_dims_array.cbegin(), C_dims_array.cend(), 1, std::multiplies<int>());
    SimpleBinaryOpCUDAKernel<TIn, TOut, BinaryOperator>
        <<<CAFFE_GET_BLOCKS(size),
           CAFFE_CUDA_NUM_THREADS,
           0,
           context->hip_stream()>>>(size, op, A, B, C);
    return;
  }
  int rows;
  int cols;
  bool broadcast_1st;
  if (utils::IsRowwiseBroadcastBinaryOp(
          ndim,
          A_dims_array.data(),
          B_dims_array.data(),
          &rows,
          &cols,
          &broadcast_1st)) {
    BinaryOpWith2DBroadcasting<TIn, TOut, BinaryOperator>(
        rows, cols, true, broadcast_1st, op, A, B, C, context);
    return;
  }
  if (utils::IsColwiseBroadcastBinaryOp(
          ndim,
          A_dims_array.data(),
          B_dims_array.data(),
          &rows,
          &cols,
          &broadcast_1st)) {
    BinaryOpWith2DBroadcasting<TIn, TOut, BinaryOperator>(
        rows, cols, false, broadcast_1st, op, A, B, C, context);
    return;
  }
  DISPATCH_FUNCTION_BY_VALUE_WITH_TYPE_3(
      ndim,
      BroadcastBinaryOpImpl,
      TIn,
      TOut,
      BinaryOperator,
      A_dims_array.data(),
      B_dims_array.data(),
      C_dims_array.data(),
      op,
      A,
      B,
      C,
      context);
}

} // namespace

#define DELEGATE_SIMPLE_CUDA_UNARY_FUNCTION(T, Func, op)            \
  __global__ void Func##CUDAKernel(const int N, const T* X, T* Y) { \
    CUDA_1D_KERNEL_LOOP(i, N) {                                     \
      Y[i] = op(X[i]);                                              \
    }                                                               \
  }                                                                 \
  template <>                                                       \
  CAFFE2_CUDA_EXPORT void Func<T, HIPContext>(                     \
      const int N, const T* x, T* y, HIPContext* context) {        \
    Func##CUDAKernel<<<                                             \
        CAFFE_GET_BLOCKS(N),                                        \
        CAFFE_CUDA_NUM_THREADS,                                     \
        0,                                                          \
        context->hip_stream()>>>(N, x, y);                         \
  }

DELEGATE_SIMPLE_CUDA_UNARY_FUNCTION(float, Exp, expf)
DELEGATE_SIMPLE_CUDA_UNARY_FUNCTION(float, Log, logf)
DELEGATE_SIMPLE_CUDA_UNARY_FUNCTION(float, Cos, cosf)
DELEGATE_SIMPLE_CUDA_UNARY_FUNCTION(float, Acos, acosf)
DELEGATE_SIMPLE_CUDA_UNARY_FUNCTION(float, Sin, sinf)
DELEGATE_SIMPLE_CUDA_UNARY_FUNCTION(float, Asin, asinf)
DELEGATE_SIMPLE_CUDA_UNARY_FUNCTION(float, Tan, tanf)
DELEGATE_SIMPLE_CUDA_UNARY_FUNCTION(float, Atan, atanf)
DELEGATE_SIMPLE_CUDA_UNARY_FUNCTION(float, Sinh, sinhf)
DELEGATE_SIMPLE_CUDA_UNARY_FUNCTION(float, Cosh, coshf)
DELEGATE_SIMPLE_CUDA_UNARY_FUNCTION(float, Tanh, tanhf)
DELEGATE_SIMPLE_CUDA_UNARY_FUNCTION(float, Abs, fabsf)
DELEGATE_SIMPLE_CUDA_UNARY_FUNCTION(float, Sqr, utils::Square<float>)
DELEGATE_SIMPLE_CUDA_UNARY_FUNCTION(float, Sqrt, sqrtf)
DELEGATE_SIMPLE_CUDA_UNARY_FUNCTION(float, Rsqrt, rsqrtf)
DELEGATE_SIMPLE_CUDA_UNARY_FUNCTION(float, Cbrt, cbrtf)

DELEGATE_SIMPLE_CUDA_UNARY_FUNCTION(float, Cube, utils::Cube<float>)
DELEGATE_SIMPLE_CUDA_UNARY_FUNCTION(double, Cube, utils::Cube<double>)
DELEGATE_SIMPLE_CUDA_UNARY_FUNCTION(
    std::int32_t,
    Cube,
    utils::Cube<std::int32_t>)
DELEGATE_SIMPLE_CUDA_UNARY_FUNCTION(
    std::int64_t,
    Cube,
    utils::Cube<std::int64_t>)

DELEGATE_SIMPLE_CUDA_UNARY_FUNCTION(bool, Not, utils::Not)

DELEGATE_SIMPLE_CUDA_UNARY_FUNCTION(float, Neg, utils::Negate<float>)
DELEGATE_SIMPLE_CUDA_UNARY_FUNCTION(double, Neg, utils::Negate<double>)
DELEGATE_SIMPLE_CUDA_UNARY_FUNCTION(
    std::int32_t,
    Neg,
    utils::Negate<std::int32_t>)
DELEGATE_SIMPLE_CUDA_UNARY_FUNCTION(
    std::int64_t,
    Neg,
    utils::Negate<std::int64_t>)

DELEGATE_SIMPLE_CUDA_UNARY_FUNCTION(float, Sign, utils::Sign<float>)
DELEGATE_SIMPLE_CUDA_UNARY_FUNCTION(double, Sign, utils::Sign<double>)
DELEGATE_SIMPLE_CUDA_UNARY_FUNCTION(
    std::int32_t,
    Sign,
    utils::Sign<std::int32_t>)
DELEGATE_SIMPLE_CUDA_UNARY_FUNCTION(
    std::int64_t,
    Sign,
    utils::Sign<std::int64_t>)

DELEGATE_SIMPLE_CUDA_UNARY_FUNCTION(float, Inv, utils::Inv<float>)
DELEGATE_SIMPLE_CUDA_UNARY_FUNCTION(double, Inv, utils::Inv<double>)

#undef DELEGATE_SIMPLE_CUDA_UNARY_FUNCTION

#define CAFFE2_SPECIALIZED_CUDA_SINCOS(T)                            \
  template <>                                                        \
  CAFFE2_CUDA_EXPORT void SinCos<T, HIPContext>(                    \
      const int N, const T* x, T* ys, T* yc, HIPContext* context) { \
    SinCosCUDAKernel<<<                                              \
        CAFFE_GET_BLOCKS(N),                                         \
        CAFFE_CUDA_NUM_THREADS,                                      \
        0,                                                           \
        context->hip_stream()>>>(N, x, ys, yc);                     \
  }
CAFFE2_SPECIALIZED_CUDA_SINCOS(float)
CAFFE2_SPECIALIZED_CUDA_SINCOS(double)
#undef CAFFE2_SPECIALIZED_CUDA_SINCOS

#define DELEGATE_SIMPLE_CUDA_BINARY_FUNCTION(TIn, TOut, Func, Op) \
  template <>                                                     \
  CAFFE2_CUDA_EXPORT void Func<TIn, HIPContext>(                 \
      const int N,                                                \
      const TIn* A,                                               \
      const TIn* B,                                               \
      TOut* C,                                                    \
      HIPContext* context) {                                     \
    SimpleBinaryOpCUDAKernel<TIn, TOut, Op<TIn>>                  \
        <<<CAFFE_GET_BLOCKS(N),                                   \
           CAFFE_CUDA_NUM_THREADS,                                \
           0,                                                     \
           context->hip_stream()>>>(N, Op<TIn>(), A, B, C);      \
  }

#define DEFINE_SIMPLE_CUDA_COMPARE_FUNCTION(Func, Op)                \
  DELEGATE_SIMPLE_CUDA_BINARY_FUNCTION(std::int32_t, bool, Func, Op) \
  DELEGATE_SIMPLE_CUDA_BINARY_FUNCTION(std::int64_t, bool, Func, Op) \
  DELEGATE_SIMPLE_CUDA_BINARY_FUNCTION(float, bool, Func, Op)        \
  DELEGATE_SIMPLE_CUDA_BINARY_FUNCTION(double, bool, Func, Op)       \
  DELEGATE_SIMPLE_CUDA_BINARY_FUNCTION(bool, bool, Func, Op)

DEFINE_SIMPLE_CUDA_COMPARE_FUNCTION(EQ, thrust::equal_to)
DEFINE_SIMPLE_CUDA_COMPARE_FUNCTION(NE, thrust::not_equal_to)
DEFINE_SIMPLE_CUDA_COMPARE_FUNCTION(LT, thrust::less)
DEFINE_SIMPLE_CUDA_COMPARE_FUNCTION(LE, thrust::less_equal)
DEFINE_SIMPLE_CUDA_COMPARE_FUNCTION(GT, thrust::greater)
DEFINE_SIMPLE_CUDA_COMPARE_FUNCTION(GE, thrust::greater_equal)

#undef DEFINE_SIMPLE_CUDA_COMPARE_FUNCTION

#define DEFINE_SIMPLE_CUDA_BINARY_FUNCTION(Func, Op)                         \
  DELEGATE_SIMPLE_CUDA_BINARY_FUNCTION(std::int32_t, std::int32_t, Func, Op) \
  DELEGATE_SIMPLE_CUDA_BINARY_FUNCTION(std::int64_t, std::int64_t, Func, Op) \
  DELEGATE_SIMPLE_CUDA_BINARY_FUNCTION(float, float, Func, Op)               \
  DELEGATE_SIMPLE_CUDA_BINARY_FUNCTION(double, double, Func, Op)             \
  DELEGATE_SIMPLE_CUDA_BINARY_FUNCTION(at::Half, at::Half, Func, Op)

DEFINE_SIMPLE_CUDA_BINARY_FUNCTION(Add, AddFunctor)
DEFINE_SIMPLE_CUDA_BINARY_FUNCTION(Sub, SubFunctor)
DEFINE_SIMPLE_CUDA_BINARY_FUNCTION(Mul, MulFunctor)
DEFINE_SIMPLE_CUDA_BINARY_FUNCTION(Div, DivFunctor)

#undef DEFINE_SIMPLE_CUDA_BINARY_FUNCTION

DELEGATE_SIMPLE_CUDA_BINARY_FUNCTION(bool, bool, And, thrust::logical_and)
DELEGATE_SIMPLE_CUDA_BINARY_FUNCTION(bool, bool, Or, thrust::logical_or)
DELEGATE_SIMPLE_CUDA_BINARY_FUNCTION(bool, bool, Xor, thrust::bit_xor)

#define DEFINE_SIMPLE_CUDA_BITWISE_BINARY_FUNCTION(Func, Op)                 \
  DELEGATE_SIMPLE_CUDA_BINARY_FUNCTION(bool, bool, Func, Op)                 \
  DELEGATE_SIMPLE_CUDA_BINARY_FUNCTION(std::int32_t, std::int32_t, Func, Op) \
  DELEGATE_SIMPLE_CUDA_BINARY_FUNCTION(std::int64_t, std::int64_t, Func, Op)

DEFINE_SIMPLE_CUDA_BITWISE_BINARY_FUNCTION(BitwiseAnd, thrust::bit_and)
DEFINE_SIMPLE_CUDA_BITWISE_BINARY_FUNCTION(BitwiseOr, thrust::bit_or)
DEFINE_SIMPLE_CUDA_BITWISE_BINARY_FUNCTION(BitwiseXor, thrust::bit_xor)

#undef DEFINE_SIMPLE_CUDA_BITWISE_BINARY_FUNCTION

DELEGATE_SIMPLE_CUDA_BINARY_FUNCTION(
    float,
    float,
    ElemwiseMax,
    thrust::maximum);

#undef DELEGATE_SIMPLE_CUDA_BINARY_FUNCTION

#define DELEGATE_2D_BROADCAST_CUDA_BINARY_FUNCTION(TIn, TOut, Func, Op)   \
  template <>                                                             \
  CAFFE2_CUDA_EXPORT void Rowwise##Func<TIn, HIPContext, true>(          \
      const int rows,                                                     \
      const int cols,                                                     \
      const TIn* A,                                                       \
      const TIn* B,                                                       \
      TOut* C,                                                            \
      HIPContext* context) {                                             \
    if (rows == 0 || cols == 0) {                                         \
      return;                                                             \
    }                                                                     \
    const int size = rows * cols;                                         \
    const FIXED_DIVISOR cols_div(cols);                                   \
    RowwiseBinaryOpCUDAKenel<TIn, TOut, Op<TIn>, true>                    \
        <<<CAFFE_GET_BLOCKS(size),                                        \
           CAFFE_CUDA_NUM_THREADS,                                        \
           0,                                                             \
           context->hip_stream()>>>(size, cols_div, Op<TIn>(), A, B, C); \
  }                                                                       \
  template <>                                                             \
  CAFFE2_CUDA_EXPORT void Rowwise##Func<TIn, HIPContext, false>(         \
      const int rows,                                                     \
      const int cols,                                                     \
      const TIn* A,                                                       \
      const TIn* B,                                                       \
      TOut* C,                                                            \
      HIPContext* context) {                                             \
    if (rows == 0 || cols == 0) {                                         \
      return;                                                             \
    }                                                                     \
    const int size = rows * cols;                                         \
    const FIXED_DIVISOR cols_div(cols);                                   \
    RowwiseBinaryOpCUDAKenel<TIn, TOut, Op<TIn>, false>                   \
        <<<CAFFE_GET_BLOCKS(size),                                        \
           CAFFE_CUDA_NUM_THREADS,                                        \
           0,                                                             \
           context->hip_stream()>>>(size, cols_div, Op<TIn>(), A, B, C); \
  }                                                                       \
  template <>                                                             \
  CAFFE2_CUDA_EXPORT void Colwise##Func<TIn, HIPContext, true>(          \
      const int rows,                                                     \
      const int cols,                                                     \
      const TIn* A,                                                       \
      const TIn* B,                                                       \
      TOut* C,                                                            \
      HIPContext* context) {                                             \
    if (rows == 0 || cols == 0) {                                         \
      return;                                                             \
    }                                                                     \
    const int size = rows * cols;                                         \
    const FIXED_DIVISOR cols_div(cols);                                   \
    ColwiseBinaryOpCUDAKenel<TIn, TOut, Op<TIn>, true>                    \
        <<<CAFFE_GET_BLOCKS(size),                                        \
           CAFFE_CUDA_NUM_THREADS,                                        \
           0,                                                             \
           context->hip_stream()>>>(size, cols_div, Op<TIn>(), A, B, C); \
  }                                                                       \
  template <>                                                             \
  CAFFE2_CUDA_EXPORT void Colwise##Func<TIn, HIPContext, false>(         \
      const int rows,                                                     \
      const int cols,                                                     \
      const TIn* A,                                                       \
      const TIn* B,                                                       \
      TOut* C,                                                            \
      HIPContext* context) {                                             \
    if (rows == 0 || cols == 0) {                                         \
      return;                                                             \
    }                                                                     \
    const int size = rows * cols;                                         \
    const FIXED_DIVISOR cols_div(cols);                                   \
    ColwiseBinaryOpCUDAKenel<TIn, TOut, Op<TIn>, false>                   \
        <<<CAFFE_GET_BLOCKS(size),                                        \
           CAFFE_CUDA_NUM_THREADS,                                        \
           0,                                                             \
           context->hip_stream()>>>(size, cols_div, Op<TIn>(), A, B, C); \
  }

#define DEFINE_2D_BROADCAST_CUDA_COMPARE_FUNCTION(Func, Op)                \
  DELEGATE_2D_BROADCAST_CUDA_BINARY_FUNCTION(std::int32_t, bool, Func, Op) \
  DELEGATE_2D_BROADCAST_CUDA_BINARY_FUNCTION(std::int64_t, bool, Func, Op) \
  DELEGATE_2D_BROADCAST_CUDA_BINARY_FUNCTION(float, bool, Func, Op)        \
  DELEGATE_2D_BROADCAST_CUDA_BINARY_FUNCTION(double, bool, Func, Op)       \
  DELEGATE_2D_BROADCAST_CUDA_BINARY_FUNCTION(bool, bool, Func, Op)

DEFINE_2D_BROADCAST_CUDA_COMPARE_FUNCTION(EQ, thrust::equal_to)
DEFINE_2D_BROADCAST_CUDA_COMPARE_FUNCTION(NE, thrust::not_equal_to)
DEFINE_2D_BROADCAST_CUDA_COMPARE_FUNCTION(LT, thrust::less)
DEFINE_2D_BROADCAST_CUDA_COMPARE_FUNCTION(LE, thrust::less_equal)
DEFINE_2D_BROADCAST_CUDA_COMPARE_FUNCTION(GT, thrust::greater)
DEFINE_2D_BROADCAST_CUDA_COMPARE_FUNCTION(GE, thrust::greater_equal)

#undef DEFINE_2D_BROADCAST_CUDA_COMPARE_FUNCTION

#define DEFINE_2D_BROADCAST_CUDA_BINARY_FUNCTION(Func, Op)             \
  DELEGATE_2D_BROADCAST_CUDA_BINARY_FUNCTION(                          \
      std::int32_t, std::int32_t, Func, Op)                            \
  DELEGATE_2D_BROADCAST_CUDA_BINARY_FUNCTION(                          \
      std::int64_t, std::int64_t, Func, Op)                            \
  DELEGATE_2D_BROADCAST_CUDA_BINARY_FUNCTION(float, float, Func, Op)   \
  DELEGATE_2D_BROADCAST_CUDA_BINARY_FUNCTION(double, double, Func, Op) \
  DELEGATE_2D_BROADCAST_CUDA_BINARY_FUNCTION(at::Half, at::Half, Func, Op)

DEFINE_2D_BROADCAST_CUDA_BINARY_FUNCTION(Add, AddFunctor)
DEFINE_2D_BROADCAST_CUDA_BINARY_FUNCTION(Sub, SubFunctor)
DEFINE_2D_BROADCAST_CUDA_BINARY_FUNCTION(Mul, MulFunctor)
DEFINE_2D_BROADCAST_CUDA_BINARY_FUNCTION(Div, DivFunctor)

#undef DEFINE_2D_BROADCAST_CUDA_BINARY_FUNCTION

DELEGATE_2D_BROADCAST_CUDA_BINARY_FUNCTION(bool, bool, And, thrust::logical_and)
DELEGATE_2D_BROADCAST_CUDA_BINARY_FUNCTION(bool, bool, Or, thrust::logical_or)
DELEGATE_2D_BROADCAST_CUDA_BINARY_FUNCTION(bool, bool, Xor, thrust::bit_xor)

#define DEFINE_2D_BROADCAST_CUDA_BITWISE_BINARY_FUNCTION(Func, Op) \
  DELEGATE_2D_BROADCAST_CUDA_BINARY_FUNCTION(bool, bool, Func, Op) \
  DELEGATE_2D_BROADCAST_CUDA_BINARY_FUNCTION(                      \
      std::int32_t, std::int32_t, Func, Op)                        \
  DELEGATE_2D_BROADCAST_CUDA_BINARY_FUNCTION(                      \
      std::int64_t, std::int64_t, Func, Op)

DEFINE_2D_BROADCAST_CUDA_BITWISE_BINARY_FUNCTION(BitwiseAnd, thrust::bit_and)
DEFINE_2D_BROADCAST_CUDA_BITWISE_BINARY_FUNCTION(BitwiseOr, thrust::bit_or)
DEFINE_2D_BROADCAST_CUDA_BITWISE_BINARY_FUNCTION(BitwiseXor, thrust::bit_xor)

#undef DEFINE_2D_BROADCAST_CUDA_BITWISE_BINARY_FUNCTION

#undef DELEGATE_2D_BROADCAST_CUDA_BINARY_FUNCTION

#define DELEGATE_BROADCAST_CUDA_BINARY_FUNCTION(TIn, TOut, Func, Op)  \
  template <>                                                         \
  CAFFE2_CUDA_EXPORT void Func<TIn, HIPContext>(                     \
      const int A_ndim,                                               \
      const int* A_dims,                                              \
      const int B_ndim,                                               \
      const int* B_dims,                                              \
      const TIn* A,                                                   \
      const TIn* B,                                                   \
      TOut* C,                                                        \
      HIPContext* context) {                                         \
    BroadcastBinaryOp<TIn, TOut, Op<TIn>>(                            \
        A_ndim, A_dims, B_ndim, B_dims, Op<TIn>(), A, B, C, context); \
  }

#define DEFINE_BROADCAST_CUDA_COMPARE_FUNCTION(Func, Op)                \
  DELEGATE_BROADCAST_CUDA_BINARY_FUNCTION(std::int32_t, bool, Func, Op) \
  DELEGATE_BROADCAST_CUDA_BINARY_FUNCTION(std::int64_t, bool, Func, Op) \
  DELEGATE_BROADCAST_CUDA_BINARY_FUNCTION(float, bool, Func, Op)        \
  DELEGATE_BROADCAST_CUDA_BINARY_FUNCTION(double, bool, Func, Op)       \
  DELEGATE_BROADCAST_CUDA_BINARY_FUNCTION(bool, bool, Func, Op)

DEFINE_BROADCAST_CUDA_COMPARE_FUNCTION(EQ, thrust::equal_to)
DEFINE_BROADCAST_CUDA_COMPARE_FUNCTION(NE, thrust::not_equal_to)
DEFINE_BROADCAST_CUDA_COMPARE_FUNCTION(LT, thrust::less)
DEFINE_BROADCAST_CUDA_COMPARE_FUNCTION(LE, thrust::less_equal)
DEFINE_BROADCAST_CUDA_COMPARE_FUNCTION(GT, thrust::greater)
DEFINE_BROADCAST_CUDA_COMPARE_FUNCTION(GE, thrust::greater_equal)

#undef DEFINE_BROADCAST_CUDA_COMPARE_FUNCTION

#define DEFINE_BROADCAST_CUDA_BINARY_FUNCTION(Func, Op)             \
  DELEGATE_BROADCAST_CUDA_BINARY_FUNCTION(                          \
      std::int32_t, std::int32_t, Func, Op)                         \
  DELEGATE_BROADCAST_CUDA_BINARY_FUNCTION(                          \
      std::int64_t, std::int64_t, Func, Op)                         \
  DELEGATE_BROADCAST_CUDA_BINARY_FUNCTION(float, float, Func, Op)   \
  DELEGATE_BROADCAST_CUDA_BINARY_FUNCTION(double, double, Func, Op) \
  DELEGATE_BROADCAST_CUDA_BINARY_FUNCTION(at::Half, at::Half, Func, Op)

DEFINE_BROADCAST_CUDA_BINARY_FUNCTION(Add, AddFunctor)
DEFINE_BROADCAST_CUDA_BINARY_FUNCTION(Sub, SubFunctor)
DEFINE_BROADCAST_CUDA_BINARY_FUNCTION(Mul, MulFunctor)
DEFINE_BROADCAST_CUDA_BINARY_FUNCTION(Div, DivFunctor)

#undef DEFINE_BROADCAST_CUDA_BINARY_FUNCTION

DELEGATE_BROADCAST_CUDA_BINARY_FUNCTION(bool, bool, And, thrust::logical_and)
DELEGATE_BROADCAST_CUDA_BINARY_FUNCTION(bool, bool, Or, thrust::logical_or)
DELEGATE_BROADCAST_CUDA_BINARY_FUNCTION(bool, bool, Xor, thrust::bit_xor)

#define DEFINE_BROADCAST_CUDA_BITWISE_BINARY_FUNCTION(Func, Op) \
  DELEGATE_BROADCAST_CUDA_BINARY_FUNCTION(bool, bool, Func, Op) \
  DELEGATE_BROADCAST_CUDA_BINARY_FUNCTION(                      \
      std::int32_t, std::int32_t, Func, Op)                     \
  DELEGATE_BROADCAST_CUDA_BINARY_FUNCTION(std::int64_t, std::int64_t, Func, Op)

DEFINE_BROADCAST_CUDA_BITWISE_BINARY_FUNCTION(BitwiseAnd, thrust::bit_and)
DEFINE_BROADCAST_CUDA_BITWISE_BINARY_FUNCTION(BitwiseOr, thrust::bit_or)
DEFINE_BROADCAST_CUDA_BITWISE_BINARY_FUNCTION(BitwiseXor, thrust::bit_xor)

#undef DEFINE_BROADCAST_CUDA_BITWISE_BINARY_FUNCTION

#undef DELEGATE_BROADCAST_CUDA_BINARY_FUNCTION

#define DELEGATE_REDUCTION_FUNCTION(T, Funcname, func)                   \
  template <>                                                            \
  CAFFE2_CUDA_EXPORT void Funcname<T, HIPContext>(                      \
      const int N,                                                       \
      const T* src,                                                      \
      T* dst,                                                            \
      Tensor* scratch_ptr,                                               \
      HIPContext* context) {                                            \
    size_t memRequired = 0;                                              \
    hipcub::DeviceReduce::func(                                             \
        nullptr, memRequired, src, dst, N, context->hip_stream());      \
    auto buffer_size =                                                   \
        static_cast<int64_t>((memRequired + sizeof(T) - 1) / sizeof(T)); \
    scratch_ptr->Resize(std::vector<int64_t>{buffer_size});              \
    hipcub::DeviceReduce::func(                                             \
        static_cast<void*>(scratch_ptr->mutable_data<T>()),              \
        memRequired,                                                     \
        src,                                                             \
        dst,                                                             \
        N,                                                               \
        context->hip_stream());                                         \
  }

DELEGATE_REDUCTION_FUNCTION(float, ReduceMin, Min)
DELEGATE_REDUCTION_FUNCTION(float, ReduceMax, Max)
DELEGATE_REDUCTION_FUNCTION(int32_t, ReduceMax, Max)
DELEGATE_REDUCTION_FUNCTION(int64_t, ReduceMax, Max)

#undef DELEGATE_REDUCTION_FUNCTION

// Caffe2 gemm provides a simpler interface to the gemm functions, with the
// limitation that the data has to be contiguous in memory.
template <>
CAFFE2_CUDA_EXPORT void Gemm<float, HIPContext>(
    const CBLAS_TRANSPOSE trans_A,
    const CBLAS_TRANSPOSE trans_B,
    const int M,
    const int N,
    const int K,
    const float alpha,
    const float* A,
    const float* B,
    const float beta,
    float* C,
    HIPContext* context,
    TensorProto::DataType math_type) {
  // Note that cublas follows fortran order, so the order is different from
  // the cblas convention.
  const int lda = (trans_A == CblasNoTrans) ? K : M;
  const int ldb = (trans_B == CblasNoTrans) ? N : K;
  const hipblasOperation_t cu_trans_A =
      (trans_A == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  const hipblasOperation_t cu_trans_B =
      (trans_B == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  CUBLAS_ENFORCE(
      hipblasSetPointerMode(context->cublas_handle(), HIPBLAS_POINTER_MODE_HOST));
  CUBLAS_ENFORCE(hipblasSgemm(
      context->cublas_handle(),
      cu_trans_B,
      cu_trans_A,
      N,
      M,
      K,
      &alpha,
      B,
      ldb,
      A,
      lda,
      &beta,
      C,
      N));
}

template <>
CAFFE2_CUDA_EXPORT void Gemm<at::Half, HIPContext>(
    const CBLAS_TRANSPOSE trans_A,
    const CBLAS_TRANSPOSE trans_B,
    const int M,
    const int N,
    const int K,
    const float alpha,
    const at::Half* A,
    const at::Half* B,
    const float beta,
    at::Half* C,
    HIPContext* context,
    TensorProto::DataType math_type) {
#if defined(__HIPCC__) && !ROCBLAS_FP16
  CAFFE_THROW("HIP currently does not support FP16 yet.");
#else
  // Note that cublas follows fortran order, so the order is different from
  // the cblas convention.
  const int lda = (trans_A == CblasNoTrans) ? K : M;
  const int ldb = (trans_B == CblasNoTrans) ? N : K;
  const hipblasOperation_t cu_trans_A =
      (trans_A == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  const hipblasOperation_t cu_trans_B =
      (trans_B == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  if (math_type == TensorProto_DataType_FLOAT) {
    CUBLAS_ENFORCE(hipblasSetPointerMode(
        context->cublas_handle(), HIPBLAS_POINTER_MODE_HOST));
    CUBLAS_ENFORCE(cublasSgemmEx(
        context->cublas_handle(),
        cu_trans_B,
        cu_trans_A,
        N,
        M,
        K,
        &alpha,
        B,
        HIP_R_16F,
        ldb,
        A,
        HIP_R_16F,
        lda,
        &beta,
        C,
        HIP_R_16F,
        N));
  } else if (math_type == TensorProto_DataType_FLOAT16) {
    // convert alpha, beta from float -> __half
    const __half alpha_fp16 = at::Half(alpha);
    const __half beta_fp16 = at::Half(beta);
    // call hipblasHgemm
    CUBLAS_ENFORCE(hipblasSetPointerMode(
        context->cublas_handle(), HIPBLAS_POINTER_MODE_HOST));
    CUBLAS_ENFORCE(hipblasHgemm(
        context->cublas_handle(),
        cu_trans_B,
        cu_trans_A,
        N,
        M,
        K,
        &alpha_fp16,
        (const __half*)B,
        ldb,
        (const __half*)A,
        lda,
        &beta_fp16,
        (__half*)C,
        N));
  } else {
    // fail
    CAFFE_THROW("Unsupported math type");
  }
#endif
}

template <>
CAFFE2_CUDA_EXPORT void BiasCHW<float, HIPContext>(
    const float* bias,
    const float* bias_multiplier,
    const int bias_channels,
    const int image_size,
    float* image,
    HIPContext* context) {
  Gemm<float, HIPContext>(
      CblasNoTrans,
      CblasNoTrans,
      bias_channels,
      image_size,
      1,
      1,
      bias,
      bias_multiplier,
      1,
      image,
      context);
}

template <>
CAFFE2_CUDA_EXPORT void GemmBatched<float, HIPContext>(
    const CBLAS_TRANSPOSE trans_A,
    const CBLAS_TRANSPOSE trans_B,
    const int batch_size,
    const int M,
    const int N,
    const int K,
    const float alpha,
    const float** A,
    const float** B,
    const float beta,
    float** C,
    HIPContext* context,
    TensorProto::DataType math_type) {
#if __CUDACC_VER_MAJOR__ < 8 || defined(__HIPCC__)
  // loop over matrices in the batch
  for (int i = 0; i < batch_size; ++i) {
    Gemm<float, HIPContext>(
        trans_A,
        trans_B,
        M,
        N,
        K,
        alpha,
        A[i],
        B[i],
        beta,
        C[i],
        context,
        math_type);
  }
#else
  // Note that cublas follows fortran order, so the order is different from
  // the cblas convention.
  const int lda = (trans_A == CblasNoTrans) ? K : M;
  const int ldb = (trans_B == CblasNoTrans) ? N : K;
  const int ldc = N;
  const hipblasOperation_t cu_trans_A =
      (trans_A == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  const hipblasOperation_t cu_trans_B =
      (trans_B == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  thrust::device_vector<const float*> A_device(A, A + batch_size);
  thrust::device_vector<const float*> B_device(B, B + batch_size);
  thrust::device_vector<float*> C_device(C, C + batch_size);
  CUBLAS_ENFORCE(
      hipblasSetPointerMode(context->cublas_handle(), HIPBLAS_POINTER_MODE_HOST));
  CUBLAS_ENFORCE(hipblasSgemmBatched(
      context->cublas_handle(),
      cu_trans_B,
      cu_trans_A,
      N,
      M,
      K,
      &alpha,
      B_device.data().get(),
      ldb,
      A_device.data().get(),
      lda,
      &beta,
      C_device.data().get(),
      ldc,
      batch_size));
#endif
}

template <>
CAFFE2_CUDA_EXPORT void GemmStridedBatched<float, HIPContext>(
    const CBLAS_TRANSPOSE trans_A,
    const CBLAS_TRANSPOSE trans_B,
    const int batch_size,
    const int M,
    const int N,
    const int K,
    const float alpha,
    const float* A,
    const int A_stride,
    const float* B,
    const int B_stride,
    const float beta,
    float* C,
    const int C_stride,
    HIPContext* context,
    TensorProto::DataType math_type) {
#if __CUDACC_VER_MAJOR__ < 8 && !defined(__HIPCC__)
  // loop over matrices in the batch
  for (int i = 0; i < batch_size; ++i) {
    Gemm<float, HIPContext>(
        trans_A, trans_B, M, N, K, alpha, A, B, beta, C, context, math_type);
    A += A_stride;
    B += B_stride;
    C += C_stride;
  }
#else
  // Note that cublas follows fortran order, so the order is different from
  // the cblas convention.
  const int lda = (trans_A == CblasNoTrans) ? K : M;
  const int ldb = (trans_B == CblasNoTrans) ? N : K;
  const int ldc = N;
  const hipblasOperation_t cu_trans_A =
      (trans_A == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  const hipblasOperation_t cu_trans_B =
      (trans_B == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  CUBLAS_ENFORCE(
      hipblasSetPointerMode(context->cublas_handle(), HIPBLAS_POINTER_MODE_HOST));
  CUBLAS_ENFORCE(hipblasSgemmStridedBatched(
      context->cublas_handle(),
      cu_trans_B,
      cu_trans_A,
      N,
      M,
      K,
      &alpha,
      B,
      ldb,
      B_stride,
      A,
      lda,
      A_stride,
      &beta,
      C,
      ldc,
      C_stride,
      batch_size));
#endif
}

template <>
CAFFE2_CUDA_EXPORT void GemmBatched<at::Half, HIPContext>(
    const CBLAS_TRANSPOSE trans_A,
    const CBLAS_TRANSPOSE trans_B,
    const int batch_size,
    const int M,
    const int N,
    const int K,
    const float alpha,
    const at::Half** A,
    const at::Half** B,
    const float beta,
    at::Half** C,
    HIPContext* context,
    TensorProto::DataType math_type) {
#if defined(__HIPCC__) && !ROCBLAS_FP16
  CAFFE_THROW("HIP currently does not support FP16 yet.");
#else
#if __CUDACC_VER_MAJOR__ < 9
  // loop over matrices in the batch
  for (int i = 0; i < batch_size; ++i) {
    Gemm<at::Half, HIPContext>(
        trans_A,
        trans_B,
        M,
        N,
        K,
        alpha,
        A[i],
        B[i],
        beta,
        C[i],
        context,
        math_type);
  }
#else
  // Note that cublas follows fortran order, so the order is different from
  // the cblas convention.
  const int lda = (trans_A == CblasNoTrans) ? K : M;
  const int ldb = (trans_B == CblasNoTrans) ? N : K;
  const int ldc = N;
  const hipblasOperation_t cu_trans_A =
      (trans_A == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  const hipblasOperation_t cu_trans_B =
      (trans_B == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  if (math_type == TensorProto_DataType_FLOAT) {
#if CUDA_VERSION < 9010
    // loop over matrices in the batch
    for (int i = 0; i < batch_size; ++i) {
      Gemm<at::Half, HIPContext>(
          trans_A,
          trans_B,
          M,
          N,
          K,
          alpha,
          A[i],
          B[i],
          beta,
          C[i],
          context,
          math_type);
    }
#else
    thrust::device_vector<const void*> A_device(A, A + batch_size);
    thrust::device_vector<const void*> B_device(B, B + batch_size);
    thrust::device_vector<void*> C_device(C, C + batch_size);
    CUBLAS_ENFORCE(hipblasSetPointerMode(
        context->cublas_handle(), HIPBLAS_POINTER_MODE_HOST));
    CUBLAS_ENFORCE(hipblasGemmBatchedEx(
        context->cublas_handle(),
        cu_trans_B,
        cu_trans_A,
        N,
        M,
        K,
        &alpha,
        B_device.data().get(),
        HIP_R_16F,
        ldb,
        A_device.data().get(),
        HIP_R_16F,
        lda,
        &beta,
        C_device.data().get(),
        HIP_R_16F,
        ldc,
        batch_size,
        HIP_R_32F,
        CUBLAS_GEMM_DEFAULT_TENSOR_OP));
#endif
  } else if (math_type == TensorProto_DataType_FLOAT16) {
    // Convert alpha, beta from float -> __half
    const __half alpha_fp16 = at::Half(alpha);
    const __half beta_fp16 = at::Half(beta);
    std::vector<const __half*> A_array(batch_size);
    std::vector<const __half*> B_array(batch_size);
    std::vector<__half*> C_array(batch_size);
    for (int i = 0; i < batch_size; ++i) {
      A_array[i] = reinterpret_cast<const __half*>(A[i]);
      B_array[i] = reinterpret_cast<const __half*>(B[i]);
      C_array[i] = reinterpret_cast<__half*>(C[i]);
    }
    thrust::device_vector<const __half*> A_device(
        A_array.cbegin(), A_array.cend());
    thrust::device_vector<const __half*> B_device(
        B_array.cbegin(), B_array.cend());
    thrust::device_vector<__half*> C_device(C_array.cbegin(), C_array.cend());
    CUBLAS_ENFORCE(hipblasSetPointerMode(
        context->cublas_handle(), HIPBLAS_POINTER_MODE_HOST));
    CUBLAS_ENFORCE(hipblasHgemmBatched(
        context->cublas_handle(),
        cu_trans_B,
        cu_trans_A,
        N,
        M,
        K,
        &alpha_fp16,
        B_device.data().get(),
        ldb,
        A_device.data().get(),
        lda,
        &beta_fp16,
        C_device.data().get(),
        ldc,
        batch_size));
  } else {
    CAFFE_THROW("Unsupported math type");
  }
#endif
#endif
}

template <>
CAFFE2_CUDA_EXPORT void GemmStridedBatched<at::Half, HIPContext>(
    const CBLAS_TRANSPOSE trans_A,
    const CBLAS_TRANSPOSE trans_B,
    const int batch_size,
    const int M,
    const int N,
    const int K,
    const float alpha,
    const at::Half* A,
    const int A_stride,
    const at::Half* B,
    const int B_stride,
    const float beta,
    at::Half* C,
    const int C_stride,
    HIPContext* context,
    TensorProto::DataType math_type) {
#if defined(__HIPCC__) && !ROCBLAS_FP16
  CAFFE_THROW("HIP currently does not support FP16 yet.");
#else
#if __CUDACC_VER_MAJOR__ < 8
  // loop over matrices in the batch
  for (int i = 0; i < batch_size; ++i) {
    Gemm<at::Half, HIPContext>(
        trans_A, trans_B, M, N, K, alpha, A, B, beta, C, context, math_type);
    A += A_stride;
    B += B_stride;
    C += C_stride;
  }
#else
  // Note that cublas follows fortran order, so the order is different from
  // the cblas convention.
  const int lda = (trans_A == CblasNoTrans) ? K : M;
  const int ldb = (trans_B == CblasNoTrans) ? N : K;
  const int ldc = N;
  const hipblasOperation_t cu_trans_A =
      (trans_A == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  const hipblasOperation_t cu_trans_B =
      (trans_B == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  if (math_type == TensorProto_DataType_FLOAT) {
#if CUDA_VERSION < 9010
    // loop over matrices in the batch
    for (int i = 0; i < batch_size; ++i) {
      Gemm<at::Half, HIPContext>(
          trans_A, trans_B, M, N, K, alpha, A, B, beta, C, context, math_type);
      A += A_stride;
      B += B_stride;
      C += C_stride;
    }
#else
    CUBLAS_ENFORCE(hipblasSetPointerMode(
        context->cublas_handle(), HIPBLAS_POINTER_MODE_HOST));
    CUBLAS_ENFORCE(hipblasGemmStridedBatchedEx(
        context->cublas_handle(),
        cu_trans_B,
        cu_trans_A,
        N,
        M,
        K,
        &alpha,
        B,
        HIP_R_16F,
        ldb,
        B_stride,
        A,
        HIP_R_16F,
        lda,
        A_stride,
        &beta,
        C,
        HIP_R_16F,
        ldc,
        C_stride,
        batch_size,
        HIP_R_32F,
        CUBLAS_GEMM_DEFAULT_TENSOR_OP));
#endif
  } else if (math_type == TensorProto_DataType_FLOAT16) {
    // Convert alpha, beta from float -> __half
    const __half alpha_fp16 = at::Half(alpha);
    const __half beta_fp16 = at::Half(beta);
    CUBLAS_ENFORCE(hipblasSetPointerMode(
        context->cublas_handle(), HIPBLAS_POINTER_MODE_HOST));
    CUBLAS_ENFORCE(hipblasHgemmStridedBatched(
        context->cublas_handle(),
        cu_trans_B,
        cu_trans_A,
        N,
        M,
        K,
        &alpha_fp16,
        (const __half*)B,
        ldb,
        B_stride,
        (const __half*)A,
        lda,
        A_stride,
        &beta_fp16,
        (__half*)C,
        ldc,
        C_stride,
        batch_size));
  } else {
    CAFFE_THROW("Unsupported math type");
  }
#endif
#endif
}

#if CUDA_VERSION >= 9000

// No change, but required. Defer to default CUDA engine
template <>
CAFFE2_CUDA_EXPORT void Gemm<float, HIPContext, TensorCoreEngine>(
    const CBLAS_TRANSPOSE trans_A,
    const CBLAS_TRANSPOSE trans_B,
    const int M,
    const int N,
    const int K,
    const float alpha,
    const float* A,
    const float* B,
    const float beta,
    float* C,
    HIPContext* context,
    TensorProto::DataType math_type) {
  return Gemm<float, HIPContext>(
      trans_A, trans_B, M, N, K, alpha, A, B, beta, C, context, math_type);
}

template <>
CAFFE2_CUDA_EXPORT void Gemm<at::Half, HIPContext, TensorCoreEngine>(
    const CBLAS_TRANSPOSE trans_A,
    const CBLAS_TRANSPOSE trans_B,
    const int M,
    const int N,
    const int K,
    const float alpha,
    const at::Half* A,
    const at::Half* B,
    const float beta,
    at::Half* C,
    HIPContext* context,
    TensorProto::DataType math_type) {
  // Note that cublas follows fortran order, so the order is different from
  // the cblas convention.
  const int lda = (trans_A == CblasNoTrans) ? K : M;
  const int ldb = (trans_B == CblasNoTrans) ? N : K;
  const hipblasOperation_t cu_trans_A =
      (trans_A == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  const hipblasOperation_t cu_trans_B =
      (trans_B == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;

  // enable TensorCore for this call on this handle
  if (TensorCoreAvailable()) {
    CUBLAS_ENFORCE(
        hipblasSetMathMode(context->cublas_handle(), HIPBLAS_TENSOR_OP_MATH));
  }

  CUBLAS_ENFORCE(
      hipblasSetPointerMode(context->cublas_handle(), HIPBLAS_POINTER_MODE_HOST));
  CUBLAS_ENFORCE(hipblasGemmEx(
      context->cublas_handle(),
      cu_trans_B,
      cu_trans_A,
      N,
      M,
      K,
      &alpha,
      B,
      HIP_R_16F,
      ldb,
      A,
      HIP_R_16F,
      lda,
      &beta,
      C,
      HIP_R_16F,
      N,
      HIP_R_32F,
      CUBLAS_GEMM_DFALT_TENSOR_OP));

  // Now disable TensorCore math for subsequent calls to this handle
  if (TensorCoreAvailable()) {
    CUBLAS_ENFORCE(
        hipblasSetMathMode(context->cublas_handle(), HIPBLAS_DEFAULT_MATH));
  }
}

template <>
CAFFE2_CUDA_EXPORT void
GemmStridedBatched<float, HIPContext, TensorCoreEngine>(
    const CBLAS_TRANSPOSE trans_A,
    const CBLAS_TRANSPOSE trans_B,
    const int batch_size,
    const int M,
    const int N,
    const int K,
    const float alpha,
    const float* A,
    const int A_stride,
    const float* B,
    const int B_stride,
    const float beta,
    float* C,
    const int C_stride,
    HIPContext* context,
    TensorProto::DataType math_type) {
  return GemmStridedBatched<float, HIPContext, DefaultEngine>(
      trans_A,
      trans_B,
      batch_size,
      M,
      N,
      K,
      alpha,
      A,
      A_stride,
      B,
      B_stride,
      beta,
      C,
      C_stride,
      context,
      math_type);
}

template <>
CAFFE2_CUDA_EXPORT void
GemmStridedBatched<at::Half, HIPContext, TensorCoreEngine>(
    const CBLAS_TRANSPOSE trans_A,
    const CBLAS_TRANSPOSE trans_B,
    const int batch_size,
    const int M,
    const int N,
    const int K,
    const float alpha,
    const at::Half* A,
    const int A_stride,
    const at::Half* B,
    const int B_stride,
    const float beta,
    at::Half* C,
    const int C_stride,
    HIPContext* context,
    TensorProto::DataType math_type) {
  return GemmStridedBatched<at::Half, HIPContext, DefaultEngine>(
      trans_A,
      trans_B,
      batch_size,
      M,
      N,
      K,
      alpha,
      A,
      A_stride,
      B,
      B_stride,
      beta,
      C,
      C_stride,
      context,
      math_type);
}

#endif // CUDA_VERSION >= 9000

template <>
CAFFE2_CUDA_EXPORT void GemmEx<float, HIPContext>(
    const CBLAS_TRANSPOSE trans_A,
    const CBLAS_TRANSPOSE trans_B,
    const int M,
    const int N,
    const int K,
    const float alpha,
    const float* A,
    const int lda,
    const float* B,
    const int ldb,
    const float beta,
    float* C,
    const int ldc,
    HIPContext* context) {
  // Note that cublas follows fortran order, so the order is different from
  // the cblas convention.
  const hipblasOperation_t cu_trans_A =
      (trans_A == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  const hipblasOperation_t cu_trans_B =
      (trans_B == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  CUBLAS_ENFORCE(
      hipblasSetPointerMode(context->cublas_handle(), HIPBLAS_POINTER_MODE_HOST));
  CUBLAS_ENFORCE(hipblasSgemm(
      context->cublas_handle(),
      cu_trans_B,
      cu_trans_A,
      N,
      M,
      K,
      &alpha,
      B,
      ldb,
      A,
      lda,
      &beta,
      C,
      ldc));
}

template <>
CAFFE2_CUDA_EXPORT void Gemv<float, HIPContext>(
    const CBLAS_TRANSPOSE trans_A,
    const int M,
    const int N,
    const float alpha,
    const float* A,
    const float* x,
    const float beta,
    float* y,
    HIPContext* context,
    TensorProto::DataType math_type) {
  const hipblasOperation_t cu_trans_A =
      (trans_A == CblasNoTrans) ? HIPBLAS_OP_T : HIPBLAS_OP_N;
  CUBLAS_ENFORCE(
      hipblasSetPointerMode(context->cublas_handle(), HIPBLAS_POINTER_MODE_HOST));
  CUBLAS_ENFORCE(hipblasSgemv(
      context->cublas_handle(),
      cu_trans_A,
      N,
      M,
      &alpha,
      A,
      N,
      x,
      1,
      &beta,
      y,
      1));
}

// Batched Add variants
namespace {

template <typename T>
__global__ void AddStripedBatchKernel(
    const int N,
    const T* first,
    T* Y,
    const int stripe,
    const int batch) {
  for (int j = 0; j < batch; j++) {
    const T* x = first + j * stripe;
    CUDA_1D_KERNEL_LOOP(i, N) {
      float tmpY = convert::To<T, float>(Y[i]);
      tmpY += convert::To<T, float>(x[i]);
      Y[i] = convert::To<float, T>(tmpY);
    }
  }
}
} // namespace

#define CAFFE2_SPECIALIZED_CUDA_ADD_STRIPED_BATCH(T)              \
  template <>                                                     \
  CAFFE2_CUDA_EXPORT void AddStripedBatch<T, HIPContext>(        \
      const int N,                                                \
      const T* first,                                             \
      T* Y,                                                       \
      const int stripe,                                           \
      const int batch,                                            \
      HIPContext* context) {                                     \
    AddStripedBatchKernel<T>                                      \
        <<<CAFFE_GET_BLOCKS(N),                                   \
           CAFFE_CUDA_NUM_THREADS,                                \
           0,                                                     \
           context->hip_stream()>>>(N, first, Y, stripe, batch); \
  }

CAFFE2_SPECIALIZED_CUDA_ADD_STRIPED_BATCH(float);
CAFFE2_SPECIALIZED_CUDA_ADD_STRIPED_BATCH(at::Half);
#undef CAFFE2_SPECIALIZED_CUDA_ADD_STRIPED_BATCH

template <>
CAFFE2_CUDA_EXPORT void Gemv<at::Half, HIPContext>(
    const CBLAS_TRANSPOSE trans_A,
    const int M,
    const int N,
    const float alpha,
    const at::Half* A,
    const at::Half* x,
    const float beta,
    at::Half* y,
    HIPContext* context,
    TensorProto::DataType math_type) {
#if defined(__HIPCC__) && !ROCBLAS_FP16
  CAFFE_THROW("HIP currently does not support FP16 yet.");
#else
  const hipblasOperation_t cu_trans_A =
      (trans_A == CblasNoTrans) ? HIPBLAS_OP_T : HIPBLAS_OP_N;

  // sort out what we need to call cublasSgemmEx / hipblasHgemm
  const int m = (cu_trans_A == HIPBLAS_OP_N) ? N : M;
  const int k = (cu_trans_A == HIPBLAS_OP_N) ? M : N;
  const int lda = (cu_trans_A == HIPBLAS_OP_N) ? m : k;
  const int ldc = m;

  if (math_type == TensorProto_DataType_FLOAT) {
    CUBLAS_ENFORCE(hipblasSetPointerMode(
        context->cublas_handle(), HIPBLAS_POINTER_MODE_HOST));
    CUBLAS_ENFORCE(cublasSgemmEx(
        context->cublas_handle(),
        cu_trans_A,
        HIPBLAS_OP_N,
        m,
        1,
        k,
        &alpha,
        A,
        HIP_R_16F,
        lda,
        x,
        HIP_R_16F,
        k,
        &beta,
        y,
        HIP_R_16F,
        ldc));
  } else if (math_type == TensorProto_DataType_FLOAT16) {
    const __half alpha_fp16 = at::Half(alpha);
    const __half beta_fp16 = at::Half(beta);
    CUBLAS_ENFORCE(hipblasSetPointerMode(
        context->cublas_handle(), HIPBLAS_POINTER_MODE_HOST));
    CUBLAS_ENFORCE(hipblasHgemm(
        context->cublas_handle(),
        cu_trans_A,
        HIPBLAS_OP_N,
        m,
        1,
        k,
        &alpha_fp16,
        (const __half*)A,
        lda,
        (const __half*)x,
        k,
        &beta_fp16,
        (__half*)y,
        ldc));
  } else {
    // fail
    CAFFE_THROW("Unsupported math type");
  }
#endif
}

namespace {

template <typename T>
__global__ void SetKernel(const int N, const T alpha, T* Y) {
  CUDA_1D_KERNEL_LOOP(i, N) {
    Y[i] = alpha;
  }
}

} // namespace

#define CAFFE2_SPECIALIZED_CUDA_SET(T)                              \
  template <>                                                       \
  CAFFE2_CUDA_API void Set<T, HIPContext>(                         \
      const size_t N, const T alpha, T* Y, HIPContext* context) {  \
    if (N == 0) {                                                   \
      return;                                                       \
    }                                                               \
    if (alpha == T(0)) {                                            \
      hipMemsetAsync(Y, 0, sizeof(T) * N, context->hip_stream()); \
    } else {                                                        \
      SetKernel<T>                                                  \
          <<<CAFFE_GET_BLOCKS(N),                                   \
             CAFFE_CUDA_NUM_THREADS,                                \
             0,                                                     \
             context->hip_stream()>>>(N, alpha, Y);                \
    }                                                               \
  }
CAFFE2_SPECIALIZED_CUDA_SET(float);
CAFFE2_SPECIALIZED_CUDA_SET(double);
CAFFE2_SPECIALIZED_CUDA_SET(bool);
CAFFE2_SPECIALIZED_CUDA_SET(int8_t);
CAFFE2_SPECIALIZED_CUDA_SET(int16_t);
CAFFE2_SPECIALIZED_CUDA_SET(int);
CAFFE2_SPECIALIZED_CUDA_SET(int64_t);
CAFFE2_SPECIALIZED_CUDA_SET(char);
CAFFE2_SPECIALIZED_CUDA_SET(uint8_t);
CAFFE2_SPECIALIZED_CUDA_SET(uint16_t);
#undef CAFFE2_SPECIALIZED_CUDA_SET

template <>
CAFFE2_CUDA_EXPORT void Set<at::Half, HIPContext>(
    const size_t N,
    const at::Half alpha,
    at::Half* Y,
    HIPContext* context) {
  if (N > 0) {
    SetKernel<at::Half>
        <<<CAFFE_GET_BLOCKS(N),
           CAFFE_CUDA_NUM_THREADS,
           0,
           context->hip_stream()>>>(N, alpha, Y);
  }
}

namespace {
template <typename T>
__global__ void
UniformShift(const size_t N, const float min, const float max, T* x) {
  float scale = max - min;
  CUDA_1D_KERNEL_LOOP(i, N) {
    x[i] = convert::To<float, T>(convert::To<T, float>(x[i]) * scale + min);
  }
}

__global__ void
UniformIntFit(const size_t N, const int min, const int max, unsigned int* x) {
  int* x_int = reinterpret_cast<int*>(x);
  int range = (max - min + 1);
  CUDA_1D_KERNEL_LOOP(i, N) {
    x_int[i] = min + static_cast<int>(x[i] % range);
  }
}
} // namespace

template <>
CAFFE2_CUDA_EXPORT void RandUniform<float, HIPContext>(
    const size_t n,
    const float min,
    const float max,
    float* r,
    HIPContext* context) {
  CURAND_ENFORCE(hiprandGenerateUniform(context->curand_generator(), r, n));
  UniformShift<float>
      <<<CAFFE_GET_BLOCKS(n),
         CAFFE_CUDA_NUM_THREADS,
         0,
         context->hip_stream()>>>(n, min, max, r);
}

template <>
CAFFE2_CUDA_EXPORT void RandUniform<double, HIPContext>(
    const size_t n,
    const double min,
    const double max,
    double* r,
    HIPContext* context) {
  CURAND_ENFORCE(
      hiprandGenerateUniformDouble(context->curand_generator(), r, n));
  UniformShift<double>
      <<<CAFFE_GET_BLOCKS(n),
         CAFFE_CUDA_NUM_THREADS,
         0,
         context->hip_stream()>>>(n, min, max, r);
}

template <>
CAFFE2_CUDA_EXPORT void RandUniform<int, HIPContext>(
    const size_t n,
    const int min,
    const int max,
    int* r,
    HIPContext* context) {
  CURAND_ENFORCE(hiprandGenerate(
      context->curand_generator(), reinterpret_cast<unsigned int*>(r), n));
  UniformIntFit<<<
      CAFFE_GET_BLOCKS(n),
      CAFFE_CUDA_NUM_THREADS,
      0,
      context->hip_stream()>>>(
      n, min, max, reinterpret_cast<unsigned int*>(r));
}

template <typename T>
size_t HandleOddLengthRandGaussian(
    const size_t n,
    const T mean,
    const T std,
    T* r,
    HIPContext* context) {
  if (n % 2 == 1) {
    std::default_random_engine generator;
    std::normal_distribution<T> distribution(mean, std);
    const T random_value = distribution(generator);
    Set<T, HIPContext>(1, random_value, r + (n - 1), context);
    return n - 1;
  }
  return n;
}

template <>
CAFFE2_CUDA_EXPORT void RandGaussian<float, HIPContext>(
    const size_t n,
    const float mean,
    const float std,
    float* r,
    HIPContext* context) {
  // If n is odd, we add a random Gaussian value at the end manually
  // and generate n-1 random values using hiprandGenerateNormal.
  // hiprandGenerateNormal requires n to be even.
  const size_t even_n =
      HandleOddLengthRandGaussian<float>(n, mean, std, r, context);
  CURAND_ENFORCE(
      hiprandGenerateNormal(context->curand_generator(), r, even_n, mean, std));
}

template <>
CAFFE2_CUDA_EXPORT void RandGaussian<double, HIPContext>(
    const size_t n,
    const double mean,
    const double std,
    double* r,
    HIPContext* context) {
  const size_t even_n =
      HandleOddLengthRandGaussian<double>(n, mean, std, r, context);
  CURAND_ENFORCE(hiprandGenerateNormalDouble(
      context->curand_generator(), r, even_n, mean, std));
}

template <>
CAFFE2_CUDA_EXPORT void Dot<float, HIPContext>(
    const int n,
    const float* a,
    const float* b,
    float* y,
    HIPContext* context) {
  CUBLAS_ENFORCE(hipblasSetPointerMode(
      context->cublas_handle(), HIPBLAS_POINTER_MODE_DEVICE));
  CUBLAS_ENFORCE(hipblasSdot(context->cublas_handle(), n, a, 1, b, 1, y));
}

template <>
CAFFE2_CUDA_EXPORT void Dot<at::Half, HIPContext>(
    const int n,
    const at::Half* a,
    const at::Half* b,
    at::Half* y,
    HIPContext* context) {
#if defined(__HIPCC__) && !ROCBLAS_FP16
  CAFFE_THROW("HIP currently does not support FP16 yet.");
#else
  // execute with 32-bit math
  CUBLAS_ENFORCE(hipblasSetPointerMode(
      context->cublas_handle(), HIPBLAS_POINTER_MODE_DEVICE));
  CUBLAS_ENFORCE(hipblasDotEx(
      context->cublas_handle(),
      n,
      a,
      HIP_R_16F,
      1,
      b,
      HIP_R_16F,
      1,
      y,
      HIP_R_16F,
      HIP_R_32F));
#endif
}

// A previous version of caffe2 used Thrust but it turns out that thrust
// reduction has an implicit scratch space allocation and deallocation, which
// may interfere with NCCL and create a deadlock. Hence we are using a custom
// reduction here.
#define SUM_KERNEL_NTHREADS 128
template <typename T>
__global__ void SumKernel(const int N, const T* X, T* Y, bool square) {
  const int idx = threadIdx.x;
  __shared__ float reduction_buffer[SUM_KERNEL_NTHREADS];

  reduction_buffer[idx] = 0;

  // A multilevel reduction.
  // N -> 128
  if (!square) {
    for (int i = idx; i < N; i += SUM_KERNEL_NTHREADS) {
      reduction_buffer[idx] += convert::To<T, float>(X[i]);
    }
  } else {
    for (int i = idx; i < N; i += SUM_KERNEL_NTHREADS) {
      float Xi = convert::To<T, float>(X[i]);
      reduction_buffer[idx] += Xi * Xi;
    }
  }
  __syncthreads();
  // 128 -> 32
  if (idx < 32) {
    reduction_buffer[idx] += reduction_buffer[idx + 32] +
        reduction_buffer[idx + 64] + reduction_buffer[idx + 96];
  }
  __syncthreads();
  // 32 -> 1
  if (idx == 0) {
    float tmp = 0;
    for (int i = 0; i < 32; ++i) {
      tmp += reduction_buffer[i];
    }
    *Y = convert::To<float, T>(tmp);
  }
}

// According to the benchmarks script
// caffe2/caffe2/experiments/python/device_reduce_sum_bench.py,
// device reduce is slower for N <= 10000.
#define DEVICE_REDUCE_SIZE_THRESHOLD 10000

namespace {

template <typename T>
__global__ void SumConvertKernel(float* sum, T* dest) {
  *dest = convert::To<float, T>(*sum);
}

template <typename T, typename IterT>
CAFFE2_CUDA_EXPORT void SumGenericIter(
    const int N,
    IterT it,
    T*& dest,
    HIPContext* context,
    Tensor* scratch_ptr) {
  size_t memRequired = 0;
  hipcub::DeviceReduce::Sum(
      nullptr, memRequired, it, dest, N, context->hip_stream());
  auto buffer_size =
      static_cast<int64_t>((memRequired + sizeof(T) - 1) / sizeof(T));
  if (!dest) {
    // allocate one more T at the end of scratch for dest
    scratch_ptr->Resize(std::vector<int64_t>{buffer_size + 1});
    dest = scratch_ptr->template mutable_data<T>() + buffer_size;
  } else {
    scratch_ptr->Resize(std::vector<int64_t>{buffer_size});
  }
  hipcub::DeviceReduce::Sum(
      static_cast<void*>(scratch_ptr->template mutable_data<T>()),
      memRequired,
      it,
      dest,
      N,
      context->hip_stream());
}
} // namespace

template <>
CAFFE2_CUDA_EXPORT void Sum<float, HIPContext>(
    const int N,
    const float* x,
    float* y,
    HIPContext* context,
    Tensor* scratch_ptr) {
  if (scratch_ptr && N > DEVICE_REDUCE_SIZE_THRESHOLD) {
    SumGenericIter<float>(N, x, y, context, scratch_ptr);
  } else {
    SumKernel<<<1, SUM_KERNEL_NTHREADS, 0, context->hip_stream()>>>(
        N, x, y, false);
  }
}

template <>
CAFFE2_CUDA_EXPORT void Sum<int32_t, HIPContext>(
    const int N,
    const int32_t* x,
    int32_t* y,
    HIPContext* context,
    Tensor* scratch_ptr) {
  if (scratch_ptr && N > DEVICE_REDUCE_SIZE_THRESHOLD) {
    SumGenericIter<int32_t>(N, x, y, context, scratch_ptr);
  } else {
    SumKernel<<<1, SUM_KERNEL_NTHREADS, 0, context->hip_stream()>>>(
        N, x, y, false);
  }
}

namespace {
template <typename T>
struct FloatTransform {
  inline __host__ __device__ float operator()(const T v) const {
    return convert::To<T, float>(v);
  }
};
} // namespace

#define CAFFE2_MATH_SUM_FUNC(T)                                           \
  template <>                                                             \
  CAFFE2_CUDA_EXPORT void Sum<T, HIPContext>(                            \
      const int N,                                                        \
      const T* x,                                                         \
      T* y,                                                               \
      HIPContext* context,                                               \
      Tensor* scratch_ptr) {                                              \
    if (scratch_ptr && N > DEVICE_REDUCE_SIZE_THRESHOLD) {                \
      FloatTransform<T> transform;                                        \
      hipcub::TransformInputIterator<float, FloatTransform<T>, const T*> it( \
          x, transform);                                                  \
      float* sum = nullptr;                                               \
      SumGenericIter<float>(N, it, sum, context, scratch_ptr);            \
      SumConvertKernel<<<1, 1, 0, context->hip_stream()>>>(sum, y);      \
    } else {                                                              \
      SumKernel<<<1, SUM_KERNEL_NTHREADS, 0, context->hip_stream()>>>(   \
          N, x, y, false);                                                \
    }                                                                     \
  }

CAFFE2_MATH_SUM_FUNC(at::Half)
#undef CAFFE2_MATH_SUM_FUNC

namespace {
template <typename T>
struct SqrTransform {
  inline __host__ __device__ T operator()(const T v) const {
    return v * v;
  }
};
} //  namespace

template <>
CAFFE2_CUDA_EXPORT void SumSqr<float, HIPContext>(
    const int N,
    const float* x,
    float* y,
    HIPContext* context,
    Tensor* scratch_ptr) {
  if (scratch_ptr && N > DEVICE_REDUCE_SIZE_THRESHOLD) {
    SqrTransform<float> transform;
    hipcub::TransformInputIterator<float, SqrTransform<float>, const float*> it(
        x, transform);
    SumGenericIter<float>(N, it, y, context, scratch_ptr);
  } else {
    SumKernel<<<1, SUM_KERNEL_NTHREADS, 0, context->hip_stream()>>>(
        N, x, y, true);
  }
}

#define CAFFE2_MATH_SUMSQR_FUNC(T)                                      \
  template <>                                                           \
  CAFFE2_CUDA_EXPORT void SumSqr<T, HIPContext>(                       \
      const int N,                                                      \
      const T* x,                                                       \
      T* y,                                                             \
      HIPContext* context,                                             \
      Tensor* scratch_ptr) {                                            \
    if (scratch_ptr && N > DEVICE_REDUCE_SIZE_THRESHOLD) {              \
      FloatTransform<T> float_transform;                                \
      hipcub::TransformInputIterator<float, FloatTransform<T>, const T*>   \
          float_it(x, float_transform);                                 \
      SqrTransform<float> sqr_transform;                                \
      hipcub::TransformInputIterator<                                      \
          float,                                                        \
          SqrTransform<float>,                                          \
          decltype(float_it)>                                           \
          it(float_it, sqr_transform);                                  \
      float* sum = nullptr;                                             \
      SumGenericIter<float>(N, it, sum, context, scratch_ptr);          \
      SumConvertKernel<<<1, 1, 0, context->hip_stream()>>>(sum, y);    \
    } else {                                                            \
      SumKernel<<<1, SUM_KERNEL_NTHREADS, 0, context->hip_stream()>>>( \
          N, x, y, true);                                               \
    }                                                                   \
  }

CAFFE2_MATH_SUMSQR_FUNC(at::Half)
#undef CAFFE2_MATH_SUMSQR_FUNC
#undef DEVICE_REDUCE_SIZE_THRESHOLD

namespace {
template <typename T>
__global__ void
SelectKernel(const int N, const int D, const T* x, const int* idx, T* y) {
  CUDA_1D_KERNEL_LOOP(i, N) {
    y[i] = x[i * D + idx[i]];
  }
}
} // namespace

template <>
CAFFE2_CUDA_EXPORT void Select<float, HIPContext>(
    const int N,
    const int D,
    const float* x,
    const int* idx,
    float* y,
    HIPContext* context) {
  SelectKernel<float>
      <<<CAFFE_GET_BLOCKS(N),
         CAFFE_CUDA_NUM_THREADS,
         0,
         context->hip_stream()>>>(N, D, x, idx, y);
}

template <>
CAFFE2_CUDA_EXPORT void Select<at::Half, HIPContext>(
    const int N,
    const int D,
    const at::Half* x,
    const int* idx,
    at::Half* y,
    HIPContext* context) {
  SelectKernel<at::Half>
      <<<CAFFE_GET_BLOCKS(N),
         CAFFE_CUDA_NUM_THREADS,
         0,
         context->hip_stream()>>>(N, D, x, idx, y);
}

namespace {

template <typename TAlpha, typename TData>
__global__ void
ScaleCUDAKernel(const int n, const TAlpha alpha, const TData* x, TData* y) {
  CUDA_1D_KERNEL_LOOP(i, n) {
#if __CUDA_ARCH__ >= 350 || defined(__HIPCC__)
    y[i] = __ldg(x + i) * static_cast<TData>(alpha);
#else
    y[i] = x[i] * static_cast<TData>(alpha);
#endif
  }
}

template <typename TAlpha, typename TData>
__global__ void
ScaleCUDAKernel(const int n, const TAlpha* alpha, const TData* x, TData* y) {
  CUDA_1D_KERNEL_LOOP(i, n) {
#if __CUDA_ARCH__ >= 350 || defined(__HIPCC__)
    y[i] = __ldg(x + i) * static_cast<TData>(__ldg(alpha));
#else
    y[i] = x[i] * static_cast<TData>(*alpha);
#endif
  }
}

template <typename T>
__global__ void PowKernel(const int n, const T* x, const T exponent, T* y) {
  CUDA_1D_KERNEL_LOOP(i, n) {
    y[i] = powf(x[i], exponent);
  }
}

} // namespace

template <>
CAFFE2_CUDA_EXPORT void Powx<float, HIPContext>(
    const int N,
    const float* a,
    const float b,
    float* y,
    HIPContext* context) {
  PowKernel<<<
      CAFFE_GET_BLOCKS(N),
      CAFFE_CUDA_NUM_THREADS,
      0,
      context->hip_stream()>>>(N, a, b, y);
}

#define DELEGATE_CUBLAS_SCALE_FUNCTION(TAlpha, TData, CuBLASFunc)            \
  template <>                                                                \
  CAFFE2_CUDA_EXPORT void Scale<TAlpha, TData, HIPContext>(                 \
      const int N,                                                           \
      const TAlpha alpha,                                                    \
      const TData* x,                                                        \
      TData* y,                                                              \
      HIPContext* context) {                                                \
    if (N == 0) {                                                            \
      return;                                                                \
    }                                                                        \
    if (x != y) {                                                            \
      hipMemcpyAsync(                                                       \
          y,                                                                 \
          x,                                                                 \
          sizeof(TData) * N,                                                 \
          hipMemcpyDeviceToDevice,                                          \
          context->hip_stream());                                           \
    }                                                                        \
    if (alpha != TAlpha(1)) {                                                \
      CUBLAS_ENFORCE(hipblasSetPointerMode(                                   \
          context->cublas_handle(), HIPBLAS_POINTER_MODE_HOST));              \
      CUBLAS_ENFORCE(CuBLASFunc(context->cublas_handle(), N, &alpha, y, 1)); \
    }                                                                        \
  }                                                                          \
  template <>                                                                \
  CAFFE2_CUDA_EXPORT void Scale<TAlpha, TData, HIPContext>(                 \
      const int N,                                                           \
      const TAlpha* alpha,                                                   \
      const TData* x,                                                        \
      TData* y,                                                              \
      HIPContext* context) {                                                \
    if (N == 0) {                                                            \
      return;                                                                \
    }                                                                        \
    if (x != y) {                                                            \
      hipMemcpyAsync(                                                       \
          y,                                                                 \
          x,                                                                 \
          sizeof(TData) * N,                                                 \
          hipMemcpyDeviceToDevice,                                          \
          context->hip_stream());                                           \
    }                                                                        \
    CUBLAS_ENFORCE(hipblasSetPointerMode(                                     \
        context->cublas_handle(), HIPBLAS_POINTER_MODE_DEVICE));              \
    CUBLAS_ENFORCE(CuBLASFunc(context->cublas_handle(), N, alpha, y, 1));    \
  }
DELEGATE_CUBLAS_SCALE_FUNCTION(float, float, hipblasSscal)
DELEGATE_CUBLAS_SCALE_FUNCTION(double, double, hipblasDscal)
#undef DELEGATE_CUBLAS_SCALE_FUNCTION

#define CAFFE2_SPECIALIZED_CUDA_SCALE(TAlpha, TData)         \
  template <>                                                \
  CAFFE2_CUDA_EXPORT void Scale<TAlpha, TData, HIPContext>( \
      const int N,                                           \
      const TAlpha alpha,                                    \
      const TData* x,                                        \
      TData* y,                                              \
      HIPContext* context) {                                \
    if (N == 0) {                                            \
      return;                                                \
    }                                                        \
    if (alpha == TAlpha(1)) {                                \
      if (x != y) {                                          \
        hipMemcpyAsync(                                     \
            y,                                               \
            x,                                               \
            sizeof(TData) * N,                               \
            hipMemcpyDeviceToDevice,                        \
            context->hip_stream());                         \
      }                                                      \
      return;                                                \
    }                                                        \
    ScaleCUDAKernel<TAlpha, TData>                           \
        <<<CAFFE_GET_BLOCKS(N),                              \
           CAFFE_CUDA_NUM_THREADS,                           \
           0,                                                \
           context->hip_stream()>>>(N, alpha, x, y);        \
  }                                                          \
  template <>                                                \
  CAFFE2_CUDA_EXPORT void Scale<TAlpha, TData, HIPContext>( \
      const int N,                                           \
      const TAlpha* alpha,                                   \
      const TData* x,                                        \
      TData* y,                                              \
      HIPContext* context) {                                \
    if (N == 0) {                                            \
      return;                                                \
    }                                                        \
    ScaleCUDAKernel<TAlpha, TData>                           \
        <<<CAFFE_GET_BLOCKS(N),                              \
           CAFFE_CUDA_NUM_THREADS,                           \
           0,                                                \
           context->hip_stream()>>>(N, alpha, x, y);        \
  }
CAFFE2_SPECIALIZED_CUDA_SCALE(std::int32_t, std::int32_t)
CAFFE2_SPECIALIZED_CUDA_SCALE(std::int64_t, std::int64_t)

#ifndef __HIPCC__
template <>
CAFFE2_CUDA_EXPORT void Scale<at::Half, at::Half, HIPContext>(
    const int N,
    const at::Half alpha,
    const at::Half* x,
    at::Half* y,
    HIPContext* context) {
  if (N == 0) {
    return;
  }
  if (x != y) {
    hipMemcpyAsync(
        y,
        x,
        sizeof(at::Half) * N,
        hipMemcpyDeviceToDevice,
        context->hip_stream());
  }
  CUBLAS_ENFORCE(
      hipblasSetPointerMode(context->cublas_handle(), HIPBLAS_POINTER_MODE_HOST));
  CUBLAS_ENFORCE(hipblasScalEx(
      context->cublas_handle(),
      N,
      &alpha,
      HIP_R_16F,
      y,
      HIP_R_16F,
      1,
      HIP_R_32F));
}

template <>
CAFFE2_CUDA_EXPORT void Scale<at::Half, at::Half, HIPContext>(
    const int N,
    const at::Half* alpha,
    const at::Half* x,
    at::Half* y,
    HIPContext* context) {
  if (N == 0) {
    return;
  }
  if (x != y) {
    hipMemcpyAsync(
        y,
        x,
        sizeof(at::Half) * N,
        hipMemcpyDeviceToDevice,
        context->hip_stream());
  }
  CUBLAS_ENFORCE(hipblasSetPointerMode(
      context->cublas_handle(), HIPBLAS_POINTER_MODE_DEVICE));
  CUBLAS_ENFORCE(hipblasScalEx(
      context->cublas_handle(),
      N,
      alpha,
      HIP_R_16F,
      y,
      HIP_R_16F,
      1,
      HIP_R_32F));
}

template <>
CAFFE2_CUDA_EXPORT void Scale<float, at::Half, HIPContext>(
    const int N,
    const float alpha,
    const at::Half* x,
    at::Half* y,
    HIPContext* context) {
  if (N == 0) {
    return;
  }
  if (x != y) {
    hipMemcpyAsync(
        y,
        x,
        sizeof(at::Half) * N,
        hipMemcpyDeviceToDevice,
        context->hip_stream());
  }
  if (alpha != 1.0f) {
    CUBLAS_ENFORCE(hipblasSetPointerMode(
        context->cublas_handle(), HIPBLAS_POINTER_MODE_HOST));
    CUBLAS_ENFORCE(hipblasScalEx(
        context->cublas_handle(),
        N,
        &alpha,
        HIP_R_32F,
        y,
        HIP_R_16F,
        1,
        HIP_R_32F));
  }
}

template <>
CAFFE2_CUDA_EXPORT void Scale<float, at::Half, HIPContext>(
    const int N,
    const float* alpha,
    const at::Half* x,
    at::Half* y,
    HIPContext* context) {
  if (N == 0) {
    return;
  }
  if (x != y) {
    hipMemcpyAsync(
        y,
        x,
        sizeof(at::Half) * N,
        hipMemcpyDeviceToDevice,
        context->hip_stream());
  }
  CUBLAS_ENFORCE(hipblasSetPointerMode(
      context->cublas_handle(), HIPBLAS_POINTER_MODE_DEVICE));
  CUBLAS_ENFORCE(hipblasScalEx(
      context->cublas_handle(),
      N,
      alpha,
      HIP_R_32F,
      y,
      HIP_R_16F,
      1,
      HIP_R_32F));
}

#else // __HIPCC__

namespace {
template <>
__global__ void ScaleCUDAKernel<at::Half, at::Half>(
    const int n,
    const at::Half alpha,
    const at::Half* x,
    at::Half* y) {
  CUDA_1D_KERNEL_LOOP(i, n) {
    y[i] = convert::To<float, at::Half>(
        convert::To<at::Half, float>(x[i]) *
        convert::To<at::Half, float>(alpha));
  }
}

template <>
__global__ void ScaleCUDAKernel<at::Half, at::Half>(
    const int n,
    const at::Half* alpha,
    const at::Half* x,
    at::Half* y) {
  CUDA_1D_KERNEL_LOOP(i, n) {
    y[i] = convert::To<float, at::Half>(
        convert::To<at::Half, float>(x[i]) *
        convert::To<at::Half, float>(*alpha));
  }
}

template <>
__global__ void ScaleCUDAKernel<float, at::Half>(
    const int n,
    const float alpha,
    const at::Half* x,
    at::Half* y) {
  CUDA_1D_KERNEL_LOOP(i, n) {
    y[i] = convert::To<float, at::Half>(
        convert::To<at::Half, float>(x[i]) * alpha);
  }
}

template <>
__global__ void ScaleCUDAKernel<float, at::Half>(
    const int n,
    const float* alpha,
    const at::Half* x,
    at::Half* y) {
  CUDA_1D_KERNEL_LOOP(i, n) {
    y[i] = convert::To<float, at::Half>(
        convert::To<at::Half, float>(x[i]) * (*alpha));
  }
}
} // namespace

CAFFE2_SPECIALIZED_HIP_SCALE(at::Half, at::Half)
CAFFE2_SPECIALIZED_HIP_SCALE(float, at::Half)
#endif // __HIPCC__

#undef CAFFE2_SPECIALIZED_CUDA_SCALE

template <>
CAFFE2_CUDA_EXPORT void Axpy<float, HIPContext>(
    const int N,
    const float alpha,
    const float* X,
    float* Y,
    HIPContext* context) {
  CUBLAS_ENFORCE(
      hipblasSetPointerMode(context->cublas_handle(), HIPBLAS_POINTER_MODE_HOST));
  CUBLAS_ENFORCE(hipblasSaxpy(context->cublas_handle(), N, &alpha, X, 1, Y, 1));
}

template <>
CAFFE2_CUDA_EXPORT void Axpy<double, HIPContext>(
    const int N,
    const float alpha,
    const double* X,
    double* Y,
    HIPContext* context) {
  double alpha_d{alpha};
  CUBLAS_ENFORCE(
      hipblasSetPointerMode(context->cublas_handle(), HIPBLAS_POINTER_MODE_HOST));
  CUBLAS_ENFORCE(
      hipblasDaxpy(context->cublas_handle(), N, &alpha_d, X, 1, Y, 1));
}

template <>
CAFFE2_CUDA_EXPORT void Axpy<at::Half, HIPContext>(
    const int N,
    const float alpha,
    const at::Half* X,
    at::Half* Y,
    HIPContext* context) {
#if defined(__HIPCC__) && !ROCBLAS_FP16
  CAFFE_THROW("HIP currently does not support FP16 yet.");
#else
  CUBLAS_ENFORCE(
      hipblasSetPointerMode(context->cublas_handle(), HIPBLAS_POINTER_MODE_HOST));
  CUBLAS_ENFORCE(hipblasAxpyEx(
      context->cublas_handle(),
      N,
      &alpha,
      HIP_R_32F,
      X,
      HIP_R_16F,
      1,
      Y,
      HIP_R_16F,
      1,
      HIP_R_32F));
#endif
}

template <>
CAFFE2_CUDA_EXPORT void Axpy<float, HIPContext>(
    const int N,
    const float* alpha,
    const float* X,
    float* Y,
    HIPContext* context) {
  CUBLAS_ENFORCE(hipblasSetPointerMode(
      context->cublas_handle(), HIPBLAS_POINTER_MODE_DEVICE));
  CUBLAS_ENFORCE(hipblasSaxpy(context->cublas_handle(), N, alpha, X, 1, Y, 1));
}

template <>
CAFFE2_CUDA_EXPORT void Axpy<at::Half, HIPContext>(
    const int N,
    const float* alpha,
    const at::Half* X,
    at::Half* Y,
    HIPContext* context) {
#if defined(__HIPCC__) && !ROCBLAS_FP16
  CAFFE_THROW("HIP currently does not support FP16 yet.");
#else
  CUBLAS_ENFORCE(hipblasSetPointerMode(
      context->cublas_handle(), HIPBLAS_POINTER_MODE_DEVICE));
  CUBLAS_ENFORCE(hipblasAxpyEx(
      context->cublas_handle(),
      N,
      alpha,
      HIP_R_32F,
      X,
      HIP_R_16F,
      1,
      Y,
      HIP_R_16F,
      1,
      HIP_R_32F));
#endif
}

namespace {

template <typename TCoeff, typename TData>
__global__ void AxpbyCUDAKernel(
    const int N,
    const TCoeff a,
    const TData* x,
    const TCoeff b,
    TData* y) {
  CUDA_1D_KERNEL_LOOP(i, N) {
#if __CUDA_ARCH__ >= 350
    y[i] = __ldg(x + i) * a + y[i] * b;
#else
    y[i] = x[i] * a + y[i] * b;
#endif
  }
}

template <>
__global__ void AxpbyCUDAKernel<float, at::Half>(
    const int N,
    const float a,
    const at::Half* x,
    const float b,
    at::Half* y) {
  CUDA_1D_KERNEL_LOOP(i, N) {
    y[i] = convert::To<float, at::Half>(
        convert::To<at::Half, float>(x[i]) * a +
        convert::To<at::Half, float>(y[i]) * b);
  }
}

template <typename TCoeff, typename TData>
__global__ void AxpbyCUDAKernel(
    const int N,
    const TCoeff* a,
    const TData* x,
    const TCoeff* b,
    TData* y) {
  CUDA_1D_KERNEL_LOOP(i, N) {
#if __CUDA_ARCH__ >= 350
    y[i] = __ldg(x + i) * __ldg(a) + y[i] * __ldg(b);
#else
    y[i] = x[i] * *a + y[i] * *b;
#endif
  }
}

template <>
__global__ void AxpbyCUDAKernel<float, at::Half>(
    const int N,
    const float* a,
    const at::Half* x,
    const float* b,
    at::Half* y) {
  CUDA_1D_KERNEL_LOOP(i, N) {
#if __CUDA_ARCH__ >= 350
    y[i] = convert::To<float, at::Half>(
        convert::To<at::Half, float>(x[i]) * __ldg(a) +
        convert::To<at::Half, float>(y[i]) * __ldg(b));
#else
    y[i] = convert::To<float, at::Half>(
        convert::To<at::Half, float>(x[i]) * *a +
        convert::To<at::Half, float>(y[i]) * *b);
#endif
  }
}

} // namespace

#define CAFFE2_SPECIALIZED_CUDA_AXPBY(TCoeff, TData)         \
  template <>                                                \
  CAFFE2_CUDA_EXPORT void Axpby<TCoeff, TData, HIPContext>( \
      const int n,                                           \
      const TCoeff a,                                        \
      const TData* x,                                        \
      const TCoeff b,                                        \
      TData* y,                                              \
      HIPContext* context) {                                \
    AxpbyCUDAKernel<TCoeff, TData>                           \
        <<<CAFFE_GET_BLOCKS(n),                              \
           CAFFE_CUDA_NUM_THREADS,                           \
           0,                                                \
           context->hip_stream()>>>(n, a, x, b, y);         \
  }                                                          \
  template <>                                                \
  CAFFE2_CUDA_EXPORT void Axpby<TCoeff, TData, HIPContext>( \
      const int n,                                           \
      const TCoeff* a,                                       \
      const TData* x,                                        \
      const TCoeff* b,                                       \
      TData* y,                                              \
      HIPContext* context) {                                \
    AxpbyCUDAKernel<TCoeff, TData>                           \
        <<<CAFFE_GET_BLOCKS(n),                              \
           CAFFE_CUDA_NUM_THREADS,                           \
           0,                                                \
           context->hip_stream()>>>(n, a, x, b, y);         \
  }
CAFFE2_SPECIALIZED_CUDA_AXPBY(float, float)
CAFFE2_SPECIALIZED_CUDA_AXPBY(float, at::Half)
#undef CAFFE2_SPECIALIZED_CUDA_AXPBY

namespace {

template <typename T>
__global__ void Im2ColNCHWCUDAKernel(
    const int n,
    const int input_h,
    const int input_w,
    const int kernel_h,
    const int kernel_w,
    const int dilation_h,
    const int dilation_w,
    const int pad_t,
    const int pad_l,
    const int stride_h,
    const int stride_w,
    const int output_h,
    const int output_w,
    const T* img_data,
    T* col_data) {
  CUDA_1D_KERNEL_LOOP(index, n) {
    const int w_out = index % output_w;
    const int h_index = index / output_w;
    const int h_out = h_index % output_h;
    const int channel_in = h_index / output_h;
    const int channel_out = channel_in * kernel_h * kernel_w;
    const int h_in = h_out * stride_h - pad_t;
    const int w_in = w_out * stride_w - pad_l;
    const int output_size = output_h * output_w;
    T* col_data_ptr =
        col_data + (channel_out * output_h + h_out) * output_w + w_out;
    const T* img_data_ptr =
        img_data + (channel_in * input_h + h_in) * input_w + w_in;
    int dh = 0;
    for (int i = 0; i < kernel_h; ++i) {
      int dw = 0;
      for (int j = 0; j < kernel_w; ++j) {
        const int h = h_in + dh;
        const int w = w_in + dw;
#if __CUDA_ARCH__ >= 350 || defined(__HIPCC__)
        *col_data_ptr = utils::IsAGeZeroAndALtB(h, input_h) &&
                utils::IsAGeZeroAndALtB(w, input_w)
            ? __ldg(img_data_ptr + dh * input_w + dw)
            : 0;
#else
        *col_data_ptr = utils::IsAGeZeroAndALtB(h, input_h) &&
                utils::IsAGeZeroAndALtB(w, input_w)
            ? img_data_ptr[dh * input_w + dw]
            : 0;
#endif
        col_data_ptr += output_size;
        dw += dilation_w;
      }
      dh += dilation_h;
    }
  }
}

template <typename T>
__global__ void Im2ColNHWCCUDAKernel(
    const int n,
    const int input_h,
    const int input_w,
    const int kernel_h,
    const int kernel_w,
    const int dilation_h,
    const int dilation_w,
    const int pad_t,
    const int pad_l,
    const int stride_h,
    const int stride_w,
    const int output_w,
    const int channels,
    const T* img_data,
    T* col_data) {
  CUDA_1D_KERNEL_LOOP(index, n) {
    const int channel_in = index % channels;
    const int w_out = index / channels % output_w;
    const int h_out = index / channels / output_w;
    const int h_in = h_out * stride_h - pad_t;
    const int w_in = w_out * stride_w - pad_l;
    T* col_data_ptr = col_data +
        (h_out * output_w + w_out) * channels * kernel_h * kernel_w +
        channel_in;
    int dh = 0;
    for (int i = 0; i < kernel_h; ++i) {
      int dw = 0;
      for (int j = 0; j < kernel_w; ++j) {
        const int h = h_in + dh;
        const int w = w_in + dw;
#if __CUDA_ARCH__ >= 350 || defined(__HIPCC__)
        *col_data_ptr = utils::IsAGeZeroAndALtB(h, input_h) &&
                utils::IsAGeZeroAndALtB(w, input_w)
            ? __ldg(img_data + (h * input_w + w) * channels + channel_in)
            : 0;
#else
        *col_data_ptr = utils::IsAGeZeroAndALtB(h, input_h) &&
                utils::IsAGeZeroAndALtB(w, input_w)
            ? img_data[(h * input_w + w) * channels + channel_in]
            : 0;
#endif
        col_data_ptr += channels;
        dw += dilation_w;
      }
      dh += dilation_h;
    }
  }
}

template <typename T>
__global__ void Col2ImNCHWCUDAKernel(
    const int n,
    const int input_h,
    const int input_w,
    const int patch_h,
    const int patch_w,
    const int dilation_h,
    const int dilation_w,
    const int pad_t,
    const int pad_l,
    const int stride_h,
    const int stride_w,
    const int output_h,
    const int output_w,
    const T* col_data,
    T* img_data) {
  const int dpatch_h = dilation_h * (patch_h - 1) + 1;
  const int dpatch_w = dilation_w * (patch_w - 1) + 1;

  CUDA_1D_KERNEL_LOOP(index, n) {
    T val = 0;
    const int w = index % input_w + pad_l;
    const int h = index / input_w % input_h + pad_t;
    const int c = index / (input_h * input_w);

    // compute the start and end of the output
    const int w_col_start = (w < dpatch_w) ? 0 : (w - dpatch_w) / stride_w + 1;
    const int w_col_end = min(w / stride_w + 1, output_w);
    const int h_col_start = (h < dpatch_h) ? 0 : (h - dpatch_h) / stride_h + 1;
    const int h_col_end = min(h / stride_h + 1, output_h);

    for (int h_col = h_col_start; h_col < h_col_end; ++h_col) {
      for (int w_col = w_col_start; w_col < w_col_end; ++w_col) {
        int h_k = (h - h_col * stride_h);
        int w_k = (w - w_col * stride_w);
        if (h_k % dilation_h == 0 && w_k % dilation_w == 0) {
          h_k /= dilation_h;
          w_k /= dilation_w;
          const int col_data_index =
              (((c * patch_h + h_k) * patch_w + w_k) * output_h + h_col) *
                  output_w +
              w_col;
#if __CUDA_ARCH__ >= 350 || defined(__HIPCC__)
          val += __ldg(col_data + col_data_index);
#else
          val += col_data[col_data_index];
#endif
        }
      }
    }
    img_data[index] = val;
  }
}

template <typename T>
__global__ void Col2ImNHWCCUDAKernel(
    const int n,
    const int input_w,
    const int channels,
    const int patch_h,
    const int patch_w,
    const int dilation_h,
    const int dilation_w,
    const int pad_t,
    const int pad_l,
    const int stride_h,
    const int stride_w,
    const int output_h,
    const int output_w,
    const T* col_data,
    T* img_data) {
  const int dpatch_h = dilation_h * (patch_h - 1) + 1;
  const int dpatch_w = dilation_w * (patch_w - 1) + 1;

  CUDA_1D_KERNEL_LOOP(index, n) {
    T val = 0;
    const int c = index % channels;
    const int w = index / channels % input_w + pad_l;
    const int h = index / channels / input_w + pad_t;
    // compute the start and end of the output
    const int w_col_start = (w < dpatch_w) ? 0 : (w - dpatch_w) / stride_w + 1;
    const int w_col_end = min(w / stride_w + 1, output_w);
    const int h_col_start = (h < dpatch_h) ? 0 : (h - dpatch_h) / stride_h + 1;
    const int h_col_end = min(h / stride_h + 1, output_h);
    const int channels_col = patch_h * patch_w * channels;

    for (int h_col = h_col_start; h_col < h_col_end; ++h_col) {
      for (int w_col = w_col_start; w_col < w_col_end; ++w_col) {
        int h_k = h - h_col * stride_h;
        int w_k = w - w_col * stride_w;
        if (h_k % dilation_h == 0 && w_k % dilation_w == 0) {
          h_k /= dilation_h;
          w_k /= dilation_w;
          const int c_col = (h_k * patch_w + w_k) * channels + c;
#if __CUDA_ARCH__ >= 350 || defined(__HIPCC__)
          val += __ldg(
              col_data + (h_col * output_w + w_col) * channels_col + c_col);
#else
          val += col_data[(h_col * output_w + w_col) * channels_col + c_col];
#endif
        }
      }
    }
    img_data[index] = val;
  }
}

template <typename T, int N, bool kCol2Im>
__global__ void Im2ColNdNCHWCUDAKernel(
    const int outer_size,
    const int inner_size,
    const int kernel_size,
    SimpleArray<int, N + 1> img_shape,
    SimpleArray<int, N + 1> col_shape,
    SimpleArray<int, N> kernel_shape,
    SimpleArray<int, N> stride,
    SimpleArray<int, N> dilation,
    SimpleArray<int, N> pad,
    const T* X_data,
    T* Y_data) {
  int d_offset[N];
  int d_iter[N];
  for (int i = blockIdx.x; i < outer_size; i += gridDim.x) {
    int offset_i = i;
#pragma unroll
    for (int d_i = N - 1; d_i >= 0; --d_i) {
      d_offset[d_i] = offset_i % kernel_shape.data[d_i];
      offset_i /= kernel_shape.data[d_i];
    }
    for (int j = threadIdx.x; j < inner_size; j += blockDim.x) {
      int offset_j = j;
#pragma unroll
      for (int d_i = N - 1; d_i >= 0; --d_i) {
        d_iter[d_i] = offset_j % col_shape.data[d_i + 1];
        offset_j /= col_shape.data[d_i + 1];
      }
      const int col_index = i * inner_size + j;
      int img_index = i / kernel_size;
      bool is_padding = false;
#pragma unroll
      for (int d_i = 0; d_i < N; ++d_i) {
        const int d_img = d_iter[d_i] * stride.data[d_i] - pad.data[d_i] +
            d_offset[d_i] * dilation.data[d_i];
        is_padding |= !utils::IsAGeZeroAndALtB(d_img, img_shape.data[d_i + 1]);
        img_index = img_index * img_shape.data[d_i + 1] + d_img;
      }
#if __CUDA_ARCH__ >= 350 || defined(__HIPCC__)
      if (!kCol2Im) {
        Y_data[col_index] = is_padding ? 0 : __ldg(X_data + img_index);
      } else if (!is_padding) {
        atomicAdd(Y_data + img_index, __ldg(X_data + col_index));
      }
#else
      if (!kCol2Im) {
        Y_data[col_index] = is_padding ? 0 : X_data[img_index];
      } else if (!is_padding) {
        atomicAdd(Y_data + img_index, X_data[col_index]);
      }
#endif
    }
  }
}

template <typename T, int N>
CAFFE2_CUDA_EXPORT void Im2ColNdNCHWCUDAImpl(
    const int img_size,
    const int col_size,
    const int* img_shape,
    const int* col_shape,
    const int* kernel_shape,
    const int* stride,
    const int* dilation,
    const int* pad,
    const float* img_data,
    float* col_data,
    HIPContext* context) {
  const int outer_size = col_shape[0];
  const int inner_size = col_size / outer_size;
  const int kernel_size = std::accumulate(
      kernel_shape, kernel_shape + N, 1, std::multiplies<int>());
  SimpleArray<int, N + 1> img_shape_array;
  SimpleArray<int, N + 1> col_shape_array;
  SimpleArray<int, N> kernel_shape_array;
  SimpleArray<int, N> stride_array;
  SimpleArray<int, N> dilation_array;
  SimpleArray<int, N> pad_array;
  std::memcpy(img_shape_array.data, img_shape, (N + 1) * sizeof(int));
  std::memcpy(col_shape_array.data, col_shape, (N + 1) * sizeof(int));
  std::memcpy(kernel_shape_array.data, kernel_shape, N * sizeof(int));
  std::memcpy(stride_array.data, stride, N * sizeof(int));
  std::memcpy(dilation_array.data, dilation, N * sizeof(int));
  std::memcpy(pad_array.data, pad, N * sizeof(int));
  Im2ColNdNCHWCUDAKernel<T, N, false>
      <<<std::min(outer_size, CAFFE_MAXIMUM_NUM_BLOCKS),
         CAFFE_CUDA_NUM_THREADS,
         0,
         context->hip_stream()>>>(
          outer_size,
          inner_size,
          kernel_size,
          img_shape_array,
          col_shape_array,
          kernel_shape_array,
          stride_array,
          dilation_array,
          pad_array,
          img_data,
          col_data);
}

template <typename T, int N>
CAFFE2_CUDA_EXPORT void Col2ImNdNCHWCUDAImpl(
    const int img_size,
    const int col_size,
    const int* img_shape,
    const int* col_shape,
    const int* kernel_shape,
    const int* stride,
    const int* dilation,
    const int* pad,
    const float* col_data,
    float* img_data,
    HIPContext* context) {
  const int outer_size = col_shape[0];
  const int inner_size = col_size / outer_size;
  const int kernel_size = std::accumulate(
      kernel_shape, kernel_shape + N, 1, std::multiplies<int>());
  SimpleArray<int, N + 1> img_shape_array;
  SimpleArray<int, N + 1> col_shape_array;
  SimpleArray<int, N> kernel_shape_array;
  SimpleArray<int, N> stride_array;
  SimpleArray<int, N> dilation_array;
  SimpleArray<int, N> pad_array;
  std::memcpy(img_shape_array.data, img_shape, (N + 1) * sizeof(int));
  std::memcpy(col_shape_array.data, col_shape, (N + 1) * sizeof(int));
  std::memcpy(kernel_shape_array.data, kernel_shape, N * sizeof(int));
  std::memcpy(stride_array.data, stride, N * sizeof(int));
  std::memcpy(dilation_array.data, dilation, N * sizeof(int));
  std::memcpy(pad_array.data, pad, N * sizeof(int));
  Set<T, HIPContext>(img_size, 0, img_data, context);
  Im2ColNdNCHWCUDAKernel<T, N, true>
      <<<std::min(outer_size, CAFFE_MAXIMUM_NUM_BLOCKS),
         CAFFE_CUDA_NUM_THREADS,
         0,
         context->hip_stream()>>>(
          outer_size,
          inner_size,
          kernel_size,
          img_shape_array,
          col_shape_array,
          kernel_shape_array,
          stride_array,
          dilation_array,
          pad_array,
          col_data,
          img_data);
}

} // namespace

template <>
CAFFE2_CUDA_EXPORT void Im2Col<float, HIPContext, StorageOrder::NCHW>(
    const int channels,
    const int height,
    const int width,
    const int kernel_h,
    const int kernel_w,
    const int dilation_h,
    const int dilation_w,
    const int pad_t,
    const int pad_l,
    const int pad_b,
    const int pad_r,
    const int stride_h,
    const int stride_w,
    const float* img_data,
    float* col_data,
    HIPContext* context,
    const int /* groups */) {
  const int dkernel_h = dilation_h * (kernel_h - 1) + 1;
  const int dkernel_w = dilation_w * (kernel_w - 1) + 1;
  const int output_h = (height + pad_t + pad_b - dkernel_h) / stride_h + 1;
  const int output_w = (width + pad_l + pad_r - dkernel_w) / stride_w + 1;
  const int num_kernels = channels * output_h * output_w;
  Im2ColNCHWCUDAKernel<float>
      <<<CAFFE_GET_BLOCKS(num_kernels),
         CAFFE_CUDA_NUM_THREADS,
         0,
         context->hip_stream()>>>(
          num_kernels,
          height,
          width,
          kernel_h,
          kernel_w,
          dilation_h,
          dilation_w,
          pad_t,
          pad_l,
          stride_h,
          stride_w,
          output_h,
          output_w,
          img_data,
          col_data);
}

template <>
CAFFE2_CUDA_EXPORT void Im2Col<float, HIPContext, StorageOrder::NHWC>(
    const int channels,
    const int height,
    const int width,
    const int kernel_h,
    const int kernel_w,
    const int dilation_h,
    const int dilation_w,
    const int pad_t,
    const int pad_l,
    const int pad_b,
    const int pad_r,
    const int stride_h,
    const int stride_w,
    const float* img_data,
    float* col_data,
    HIPContext* context,
    const int groups) {
  CAFFE_ENFORCE_EQ(groups, 1, "groups must be 1 for GPU NHWC Im2Col");

  const int dkernel_h = dilation_h * (kernel_h - 1) + 1;
  const int dkernel_w = dilation_w * (kernel_w - 1) + 1;
  const int output_h = (height + pad_t + pad_b - dkernel_h) / stride_h + 1;
  const int output_w = (width + pad_l + pad_r - dkernel_w) / stride_w + 1;
  const int num_kernels = output_h * output_w * channels;
  Im2ColNHWCCUDAKernel<float>
      <<<CAFFE_GET_BLOCKS(num_kernels),
         CAFFE_CUDA_NUM_THREADS,
         0,
         context->hip_stream()>>>(
          num_kernels,
          height,
          width,
          kernel_h,
          kernel_w,
          dilation_h,
          dilation_w,
          pad_t,
          pad_l,
          stride_h,
          stride_w,
          output_w,
          channels,
          img_data,
          col_data);
}

template <>
CAFFE2_CUDA_EXPORT void Col2Im<float, HIPContext, StorageOrder::NCHW>(
    const int channels,
    const int height,
    const int width,
    const int kernel_h,
    const int kernel_w,
    const int dilation_h,
    const int dilation_w,
    const int pad_t,
    const int pad_l,
    const int pad_b,
    const int pad_r,
    const int stride_h,
    const int stride_w,
    const float* col_data,
    float* img_data,
    HIPContext* context,
    const int /* groups */) {
  // In NCHW, the number of groups doesn't affect Col2Im.
  const int dkernel_h = dilation_h * (kernel_h - 1) + 1;
  const int dkernel_w = dilation_w * (kernel_w - 1) + 1;
  const int output_h = (height + pad_t + pad_b - dkernel_h) / stride_h + 1;
  const int output_w = (width + pad_l + pad_r - dkernel_w) / stride_w + 1;
  const int num_kernels = channels * height * width;
  Col2ImNCHWCUDAKernel<float>
      <<<CAFFE_GET_BLOCKS(num_kernels),
         CAFFE_CUDA_NUM_THREADS,
         0,
         context->hip_stream()>>>(
          num_kernels,
          height,
          width,
          kernel_h,
          kernel_w,
          dilation_h,
          dilation_w,
          pad_t,
          pad_l,
          stride_h,
          stride_w,
          output_h,
          output_w,
          col_data,
          img_data);
}

template <>
CAFFE2_CUDA_EXPORT void Col2Im<float, HIPContext, StorageOrder::NHWC>(
    const int channels,
    const int height,
    const int width,
    const int kernel_h,
    const int kernel_w,
    const int dilation_h,
    const int dilation_w,
    const int pad_t,
    const int pad_l,
    const int pad_b,
    const int pad_r,
    const int stride_h,
    const int stride_w,
    const float* col_data,
    float* img_data,
    HIPContext* context,
    const int groups) {
  CAFFE_ENFORCE_EQ(groups, 1, "groups must be 1 for GPU NHWC Col2Im");

  const int dkernel_h = dilation_h * (kernel_h - 1) + 1;
  const int dkernel_w = dilation_w * (kernel_w - 1) + 1;
  const int output_h = (height + pad_t + pad_b - dkernel_h) / stride_h + 1;
  const int output_w = (width + pad_l + pad_r - dkernel_w) / stride_w + 1;
  const int num_kernels = height * width * channels;
  Col2ImNHWCCUDAKernel<float>
      <<<CAFFE_GET_BLOCKS(num_kernels),
         CAFFE_CUDA_NUM_THREADS,
         0,
         context->hip_stream()>>>(
          num_kernels,
          width,
          channels,
          kernel_h,
          kernel_w,
          dilation_h,
          dilation_w,
          pad_t,
          pad_l,
          stride_h,
          stride_w,
          output_h,
          output_w,
          col_data,
          img_data);
}

template <>
CAFFE2_CUDA_EXPORT void Im2ColNd<float, HIPContext, StorageOrder::NCHW>(
    const int N,
    const int img_size,
    const int col_size,
    const int* img_shape,
    const int* col_shape,
    const int* kernel_shape,
    const int* stride,
    const int* dilation,
    const int* pad,
    const float* img_data,
    float* col_data,
    HIPContext* context,
    const int /* groups */) {
  // In NCHW, the number of groups doesn't affect Im2Col.
  DISPATCH_FUNCTION_BY_VALUE_WITH_TYPE_1(
      N,
      Im2ColNdNCHWCUDAImpl,
      float,
      img_size,
      col_size,
      img_shape,
      col_shape,
      kernel_shape,
      stride,
      dilation,
      pad,
      img_data,
      col_data,
      context);
}

template <>
CAFFE2_CUDA_EXPORT void Im2ColNd<float, HIPContext, StorageOrder::NHWC>(
    const int N,
    const int img_size,
    const int col_size,
    const int* img_shape,
    const int* col_shape,
    const int* kernel_shape,
    const int* stride,
    const int* dilation,
    const int* pad,
    const float* img_data,
    float* col_data,
    HIPContext* context,
    const int groups) {
  CAFFE_NOT_IMPLEMENTED;
}

template <>
CAFFE2_CUDA_EXPORT void Col2ImNd<float, HIPContext, StorageOrder::NCHW>(
    const int N,
    const int img_size,
    const int col_size,
    const int* img_shape,
    const int* col_shape,
    const int* kernel_shape,
    const int* stride,
    const int* dilation,
    const int* pad,
    const float* col_data,
    float* img_data,
    HIPContext* context,
    int /* groups */) {
  // In NCHW, the number of groups doesn't affect Col2Im.
  DISPATCH_FUNCTION_BY_VALUE_WITH_TYPE_1(
      N,
      Col2ImNdNCHWCUDAImpl,
      float,
      img_size,
      col_size,
      img_shape,
      col_shape,
      kernel_shape,
      stride,
      dilation,
      pad,
      col_data,
      img_data,
      context);
}

template <>
CAFFE2_CUDA_EXPORT void Col2ImNd<float, HIPContext, StorageOrder::NHWC>(
    const int N,
    const int img_size,
    const int col_size,
    const int* img_shape,
    const int* col_shape,
    const int* kernel_shape,
    const int* stride,
    const int* dilation,
    const int* pad,
    const float* col_data,
    float* img_data,
    HIPContext* context,
    int groups) {
  CAFFE_NOT_IMPLEMENTED;
}

template <>
CAFFE2_CUDA_EXPORT void CopyMatrix<HIPContext>(
    const size_t itemsize,
    const int M,
    const int N,
    const void* A,
    const int lda,
    void* B,
    const int ldb,
    HIPContext* context,
    TypeMeta::Copy copy) {
  CAFFE_ENFORCE(!copy, "Copy constructor is not supported in CUDA context");
  hipMemcpy2DAsync(
      B,
      ldb * itemsize,
      A,
      lda * itemsize,
      N * itemsize,
      M,
      hipMemcpyDeviceToDevice,
      context->hip_stream());
}

#define CAFFE2_SPECIALIZED_CUDA_COPY_MATRIX(T) \
  template <>                                  \
  void CopyMatrix<T, HIPContext>(             \
      const int M,                             \
      const int N,                             \
      const T* A,                              \
      const int lda,                           \
      T* B,                                    \
      const int ldb,                           \
      HIPContext* context) {                  \
    if (M == 0 || N == 0) {                    \
      return;                                  \
    }                                          \
    hipMemcpy2DAsync(                         \
        B,                                     \
        sizeof(T) * ldb,                       \
        A,                                     \
        sizeof(T) * lda,                       \
        sizeof(T) * N,                         \
        M,                                     \
        hipMemcpyDeviceToDevice,              \
        context->hip_stream());               \
  }
CAFFE2_SPECIALIZED_CUDA_COPY_MATRIX(float)
CAFFE2_SPECIALIZED_CUDA_COPY_MATRIX(double)
CAFFE2_SPECIALIZED_CUDA_COPY_MATRIX(int)
CAFFE2_SPECIALIZED_CUDA_COPY_MATRIX(int64_t)
#undef CAFFE2_SPECIALIZED_CUDA_COPY_MATRIX

template <>
CAFFE2_CUDA_EXPORT void CopyVector<float, HIPContext>(
    const int N,
    const float* src,
    float* dst,
    HIPContext* context) {
  if (src != dst && N > 0) {
    hipMemcpyAsync(
        dst,
        src,
        sizeof(float) * N,
        hipMemcpyDeviceToDevice,
        context->hip_stream());
  }
}

namespace {

template <typename T>
using BlockReduce = hipcub::BlockReduce<T, CAFFE_CUDA_NUM_THREADS>;

template <typename T, class Reducer>
__global__ void RowwiseReduceKernel(
    const int rows,
    const int cols,
    const Reducer reducer,
    const T init,
    const T alpha,
    const T* X,
    T* Y) {
  __shared__ typename BlockReduce<T>::TempStorage temp_storage;
  for (int i = blockIdx.x; i < rows; i += gridDim.x) {
    T val = init;
    for (int j = threadIdx.x; j < cols; j += blockDim.x) {
      val = reducer(X[i * cols + j], val);
    }
    val = BlockReduce<T>(temp_storage).Reduce(val, reducer);
    if (threadIdx.x == 0) {
      Y[i] = val * alpha;
    }
    __syncthreads();
  }
}

template <typename T, class Reducer>
__global__ void ColwiseReduceKernel(
    const int rows,
    const int cols,
    const Reducer reducer,
    const T init,
    const T alpha,
    const T* X,
    T* Y) {
  __shared__ typename BlockReduce<T>::TempStorage temp_storage;
  for (int i = blockIdx.x; i < cols; i += gridDim.x) {
    T val = init;
    for (int j = threadIdx.x; j < rows; j += blockDim.x) {
      val = reducer(X[j * cols + i], val);
    }
    val = BlockReduce<T>(temp_storage).Reduce(val, reducer);
    if (threadIdx.x == 0) {
      Y[i] = val * alpha;
    }
    __syncthreads();
  }
}

} // namespace

#define CAFFE2_SPECIALIZED_CUDA_ROWWISE_MAX(T)                            \
  template <>                                                             \
  CAFFE2_CUDA_EXPORT void RowwiseMax<T, HIPContext>(                     \
      const int N, const int D, const T* x, T* y, HIPContext* context) { \
    RowwiseReduceKernel<<<                                                \
        std::min(N, CAFFE_MAXIMUM_NUM_BLOCKS),                            \
        CAFFE_CUDA_NUM_THREADS,                                           \
        0,                                                                \
        context->hip_stream()>>>(                                        \
        N, D, hipcub::Max(), std::numeric_limits<T>::lowest(), T(1), x, y);  \
  }
CAFFE2_SPECIALIZED_CUDA_ROWWISE_MAX(float)
#undef CAFFE2_SPECIALIZED_CUDA_ROWWISE_MAX

#define CAFFE2_SPECIALIZED_CUDA_COLWISE_MAX(T)                            \
  template <>                                                             \
  CAFFE2_CUDA_EXPORT void ColwiseMax<T, HIPContext>(                     \
      const int N, const int D, const T* x, T* y, HIPContext* context) { \
    ColwiseReduceKernel<<<                                                \
        std::min(D, CAFFE_MAXIMUM_NUM_BLOCKS),                            \
        CAFFE_CUDA_NUM_THREADS,                                           \
        0,                                                                \
        context->hip_stream()>>>(                                        \
        N, D, hipcub::Max(), std::numeric_limits<T>::lowest(), T(1), x, y);  \
  }
CAFFE2_SPECIALIZED_CUDA_COLWISE_MAX(float)
#undef CAFFE2_SPECIALIZED_CUDA_COLWISE_MAX

namespace {
__global__ void
maximum_kernel(const int N, const float alpha, const float* x, float* y) {
  CUDA_1D_KERNEL_LOOP(i, N) {
    y[i] = fmaxf(x[i], alpha);
  }
}
} // namespace

template <>
CAFFE2_CUDA_EXPORT void Maximum(
    const int N,
    const float alpha,
    const float* x,
    float* y,
    HIPContext* context) {
  maximum_kernel<<<
      std::min(N, CAFFE_MAXIMUM_NUM_BLOCKS),
      CAFFE_CUDA_NUM_THREADS,
      0,
      context->hip_stream()>>>(N, alpha, x, y);
}

namespace {

template <typename T, class Reducer, int D>
__global__ void ReduceTensorCUDAKernel(
    const int outer_size,
    const int inner_size,
    SimpleArray<int, D> X_strides,
    SimpleArray<FIXED_DIVISOR, D> Y_dims,
    const Reducer reducer,
    const T init,
    const T alpha,
    const T* X,
    T* Y) {
  __shared__ typename BlockReduce<T>::TempStorage temp_storage;
  for (int i = blockIdx.x; i < outer_size; i += gridDim.x) {
    T val = init;
    for (int j = threadIdx.x; j < inner_size; j += blockDim.x) {
      int X_index = 0;
      int Y_index = i * inner_size + j;
#pragma unroll
      for (int d = D - 1; d >= 0; --d) {
        int r;
        FIXED_DIVISOR_DIV_MOD(Y_dims.data[d], Y_index, &Y_index, &r);
        X_index += r * X_strides.data[d];
      }
#if __CUDA_ARCH__ >= 350
      val = reducer(val, __ldg(X + X_index));
#else
      val = reducer(val, X[X_index]);
#endif
    }
    val = BlockReduce<T>(temp_storage).Reduce(val, reducer);
    if (threadIdx.x == 0) {
      Y[i] = val * alpha;
    }
    __syncthreads();
  }
}

template <typename T, class Reducer, int D>
CAFFE2_CUDA_EXPORT void ReduceTensorCUDAImpl(
    const int outer_size,
    const int inner_size,
    const int* dims,
    const int* axes,
    const Reducer& reducer,
    const T init,
    const T alpha,
    const T* X,
    T* Y,
    HIPContext* context) {
  SimpleArray<int, D> X_strides;
  SimpleArray<FIXED_DIVISOR, D> Y_dims;
  utils::ComputeTransposedStrides(D, dims, axes, X_strides.data);
  for (int i = 0; i < D; ++i) {
    Y_dims.data[i] = FIXED_DIVISOR(dims[axes[i]]);
  }
  ReduceTensorCUDAKernel<T, Reducer, D>
      <<<std::min(outer_size, CAFFE_MAXIMUM_NUM_BLOCKS),
         CAFFE_CUDA_NUM_THREADS,
         0,
         context->hip_stream()>>>(
          outer_size,
          inner_size,
          X_strides,
          Y_dims,
          reducer,
          init,
          alpha,
          X,
          Y);
}

template <typename T, class Reducer>
CAFFE2_CUDA_EXPORT void ReduceTensorCUDA(
    const int num_dims,
    const int* dims,
    const int num_axes,
    const int* axes,
    const Reducer& reducer,
    const T init,
    const T alpha,
    const T* X,
    T* Y,
    HIPContext* context) {
  CAFFE_ENFORCE_LE(num_axes, num_dims);
  std::vector<int> Y_dims_vector(dims, dims + num_dims);
  for (int i = 0; i < num_axes; ++i) {
    Y_dims_vector[axes[i]] = 1;
  }
  const int* X_dims = dims;
  const int* Y_dims = Y_dims_vector.data();
  const int X_size =
      std::accumulate(X_dims, X_dims + num_dims, 1, std::multiplies<int>());
  const int Y_size =
      std::accumulate(Y_dims, Y_dims + num_dims, 1, std::multiplies<int>());
  if (X_size == 0) {
    Set<T, HIPContext>(Y_size, alpha * init, Y, context);
    return;
  }
  if (alpha == T(0)) {
    Set<T, HIPContext>(Y_size, T(0), Y, context);
    return;
  }
  if (std::equal(X_dims, X_dims + num_dims, Y_dims)) {
    Scale<T, T, HIPContext>(X_size, alpha, X, Y, context);
    return;
  }
  int rows;
  int cols;
  if (utils::IsRowwiseReduce(num_dims, X_dims, Y_dims, &rows, &cols)) {
    RowwiseReduceKernel<T>
        <<<std::min(rows, CAFFE_MAXIMUM_NUM_BLOCKS),
           CAFFE_CUDA_NUM_THREADS,
           0,
           context->hip_stream()>>>(rows, cols, reducer, init, alpha, X, Y);
    return;
  }
  if (utils::IsColwiseReduce(num_dims, X_dims, Y_dims, &rows, &cols)) {
    ColwiseReduceKernel<T>
        <<<std::min(rows, CAFFE_MAXIMUM_NUM_BLOCKS),
           CAFFE_CUDA_NUM_THREADS,
           0,
           context->hip_stream()>>>(rows, cols, reducer, init, alpha, X, Y);
    return;
  }
  std::vector<int> transpose_axes(num_dims);
  utils::ComputeTransposeAxesForReduceOp(
      num_dims, num_axes, axes, transpose_axes.data());
  const int outer_size = Y_size;
  const int inner_size = X_size / Y_size;
  DISPATCH_FUNCTION_BY_VALUE_WITH_TYPE_2(
      num_dims,
      ReduceTensorCUDAImpl,
      T,
      Reducer,
      outer_size,
      inner_size,
      dims,
      transpose_axes.data(),
      reducer,
      init,
      alpha,
      X,
      Y,
      context);
}

} // namespace

#define CAFFE2_SPECIALIZED_CUDA_REDUCE_MIN(T)        \
  template <>                                        \
  CAFFE2_CUDA_EXPORT void ReduceMin<T, HIPContext>( \
      const int num_dims,                            \
      const int* dims,                               \
      const int num_axes,                            \
      const int* axes,                               \
      const T alpha,                                 \
      const T* X,                                    \
      T* Y,                                          \
      HIPContext* context) {                        \
    ReduceTensorCUDA(                                \
        num_dims,                                    \
        dims,                                        \
        num_axes,                                    \
        axes,                                        \
        hipcub::Min(),                                  \
        std::numeric_limits<T>::max(),               \
        alpha,                                       \
        X,                                           \
        Y,                                           \
        context);                                    \
  }
CAFFE2_SPECIALIZED_CUDA_REDUCE_MIN(std::int32_t)
CAFFE2_SPECIALIZED_CUDA_REDUCE_MIN(std::int64_t)
CAFFE2_SPECIALIZED_CUDA_REDUCE_MIN(float)
CAFFE2_SPECIALIZED_CUDA_REDUCE_MIN(double)
#undef CAFFE2_SPECIALIZED_CUDA_REDUCE_MIN

#define CAFFE2_SPECIALIZED_CUDA_REDUCE_MAX(T)        \
  template <>                                        \
  CAFFE2_CUDA_EXPORT void ReduceMax<T, HIPContext>( \
      const int num_dims,                            \
      const int* dims,                               \
      const int num_axes,                            \
      const int* axes,                               \
      const T alpha,                                 \
      const T* X,                                    \
      T* Y,                                          \
      HIPContext* context) {                        \
    ReduceTensorCUDA(                                \
        num_dims,                                    \
        dims,                                        \
        num_axes,                                    \
        axes,                                        \
        hipcub::Max(),                                  \
        std::numeric_limits<T>::lowest(),            \
        alpha,                                       \
        X,                                           \
        Y,                                           \
        context);                                    \
  }
CAFFE2_SPECIALIZED_CUDA_REDUCE_MAX(std::int32_t)
CAFFE2_SPECIALIZED_CUDA_REDUCE_MAX(std::int64_t)
CAFFE2_SPECIALIZED_CUDA_REDUCE_MAX(float)
CAFFE2_SPECIALIZED_CUDA_REDUCE_MAX(double)
#undef CAFFE2_SPECIALIZED_CUDA_REDUCE_MAX

#define CAFFE2_SPECIALIZED_CUDA_REDUCE_SUM(T)        \
  template <>                                        \
  CAFFE2_CUDA_EXPORT void ReduceSum<T, HIPContext>( \
      const int num_dims,                            \
      const int* dims,                               \
      const int num_axes,                            \
      const int* axes,                               \
      const T alpha,                                 \
      const T* X,                                    \
      T* Y,                                          \
      HIPContext* context) {                        \
    ReduceTensorCUDA(                                \
        num_dims,                                    \
        dims,                                        \
        num_axes,                                    \
        axes,                                        \
        hipcub::Sum(),                                  \
        T(0),                                        \
        alpha,                                       \
        X,                                           \
        Y,                                           \
        context);                                    \
  }
CAFFE2_SPECIALIZED_CUDA_REDUCE_SUM(std::int32_t)
CAFFE2_SPECIALIZED_CUDA_REDUCE_SUM(std::int64_t)
CAFFE2_SPECIALIZED_CUDA_REDUCE_SUM(float)
CAFFE2_SPECIALIZED_CUDA_REDUCE_SUM(double)
#undef CAFFE2_SPECIALIZED_CUDA_REDUCE_SUM

#define CAFFE2_SPECIALIZED_CUDA_REDUCE_MEAN(T)        \
  template <>                                         \
  CAFFE2_CUDA_EXPORT void ReduceMean<T, HIPContext>( \
      const int num_dims,                             \
      const int* dims,                                \
      const int num_axes,                             \
      const int* axes,                                \
      const T alpha,                                  \
      const T* X,                                     \
      T* Y,                                           \
      HIPContext* context) {                         \
    int scale = 1;                                    \
    for (int i = 0; i < num_axes; ++i) {              \
      scale *= dims[axes[i]];                         \
    }                                                 \
    ReduceTensorCUDA(                                 \
        num_dims,                                     \
        dims,                                         \
        num_axes,                                     \
        axes,                                         \
        hipcub::Sum(),                                   \
        T(0),                                         \
        alpha / static_cast<T>(scale),                \
        X,                                            \
        Y,                                            \
        context);                                     \
  }
CAFFE2_SPECIALIZED_CUDA_REDUCE_MEAN(float)
#undef CAFFE2_SPECIALIZED_CUDA_REDUCE_MEAN

namespace {

template <typename T, int D>
__global__ void BroadcastCUDAKernel(
    const int Y_size,
    const SimpleArray<int, D> X_strides,
    const SimpleArray<FIXED_DIVISOR, D> Y_dims,
    const T alpha,
    const T* X,
    T* Y) {
  CUDA_1D_KERNEL_LOOP(Y_index, Y_size) {
    int X_index = 0;
    int Y_index_val = Y_index;
#pragma unroll
    for (int i = D - 1; i >= 0; --i) {
      int d;
      FIXED_DIVISOR_DIV_MOD(Y_dims.data[i], Y_index_val, &Y_index_val, &d);
      X_index += d * X_strides.data[i];
    }
#if __CUDA_ARCH__ >= 350 || defined(__HIPCC__)
    Y[Y_index] = __ldg(X + X_index) * alpha;
#else
    Y[Y_index] = X[X_index] * alpha;
#endif
  }
}

template <typename T, int D>
CAFFE2_CUDA_EXPORT void BroadcastCUDAImpl(
    const int X_ndim,
    const int* X_dims,
    const int* Y_dims,
    const T alpha,
    const T* X,
    T* Y,
    HIPContext* context) {
  SimpleArray<int, D> X_strides_array;
  SimpleArray<FIXED_DIVISOR, D> Y_dims_array;
  const int d = D - X_ndim;
  std::fill(X_strides_array.data, X_strides_array.data + d, 0);
  int cur_stride = 1;
  for (int i = D - 1; i >= d; --i) {
    CAFFE_ENFORCE(X_dims[i - d] == 1 || X_dims[i - d] == Y_dims[i]);
    X_strides_array.data[i] = X_dims[i - d] == 1 ? 0 : cur_stride;
    cur_stride *= X_dims[i - d];
  }
  for (int i = 0; i < D; ++i) {
    if (Y_dims[i] == 0) {
      return;
    }
    Y_dims_array.data[i] = FIXED_DIVISOR(Y_dims[i]);
  }
  const int Y_size =
      std::accumulate(Y_dims, Y_dims + D, 1, std::multiplies<int>());
  BroadcastCUDAKernel<T, D>
      <<<CAFFE_GET_BLOCKS(Y_size),
         CAFFE_CUDA_NUM_THREADS,
         0,
         context->hip_stream()>>>(
          Y_size, X_strides_array, Y_dims_array, alpha, X, Y);
}

} // namespace

#define CAFFE2_SPECIALIZED_CUDA_BROADCAST(T)         \
  template <>                                        \
  CAFFE2_CUDA_EXPORT void Broadcast<T, HIPContext>( \
      const int X_ndim,                              \
      const int* X_dims,                             \
      const int Y_ndim,                              \
      const int* Y_dims,                             \
      const T alpha,                                 \
      const T* X,                                    \
      T* Y,                                          \
      HIPContext* context) {                        \
    CAFFE_ENFORCE_LE(X_ndim, Y_ndim);                \
    DISPATCH_FUNCTION_BY_VALUE_WITH_TYPE_1(          \
        Y_ndim,                                      \
        BroadcastCUDAImpl,                           \
        T,                                           \
        X_ndim,                                      \
        X_dims,                                      \
        Y_dims,                                      \
        alpha,                                       \
        X,                                           \
        Y,                                           \
        context);                                    \
  }
CAFFE2_SPECIALIZED_CUDA_BROADCAST(std::int32_t)
CAFFE2_SPECIALIZED_CUDA_BROADCAST(std::int64_t)
CAFFE2_SPECIALIZED_CUDA_BROADCAST(float)
CAFFE2_SPECIALIZED_CUDA_BROADCAST(double)
#undef CAFFE2_SPECIALIZED_CUDA_BROADCAST

namespace {

template <typename T>
__global__ void RowwiseMomentsCUDAKernel(
    const int rows,
    const int cols,
    const T* X,
    T* mean,
    T* variance) {
  __shared__ typename BlockReduce<T>::TempStorage m_storage;
  __shared__ typename BlockReduce<T>::TempStorage v_storage;
  const T scale = T(1) / static_cast<T>(cols);
  for (int i = blockIdx.x; i < rows; i += gridDim.x) {
    T m_val = 0;
    T v_val = 0;
    for (int j = threadIdx.x; j < cols; j += blockDim.x) {
      const int X_index = i * cols + j;
#if __CUDA_ARCH__ >= 350 || defined(__HIPCC__)
      m_val += __ldg(X + X_index);
      v_val += __ldg(X + X_index) * __ldg(X + X_index);
#else
      m_val += X[X_index];
      v_val += X[X_index] * X[X_index];
#endif
    }
    m_val = BlockReduce<T>(m_storage).Sum(m_val);
    v_val = BlockReduce<T>(v_storage).Sum(v_val);
    if (threadIdx.x == 0) {
      const T mu = m_val * scale;
      mean[i] = mu;
      variance[i] = v_val * scale - mu * mu;
    }
    __syncthreads();
  }
}

template <typename T>
__global__ void ColwiseMomentsCUDAKernel(
    const int rows,
    const int cols,
    const T* X,
    T* mean,
    T* variance) {
  __shared__ typename BlockReduce<T>::TempStorage m_storage;
  __shared__ typename BlockReduce<T>::TempStorage v_storage;
  const T scale = T(1) / static_cast<T>(rows);
  for (int i = blockIdx.x; i < cols; i += gridDim.x) {
    T m_val = 0;
    T v_val = 0;
    for (int j = threadIdx.x; j < rows; j += blockDim.x) {
      const int X_index = j * cols + i;
#if __CUDA_ARCH__ >= 350 || defined(__HIPCC__)
      m_val += __ldg(X + X_index);
      v_val += __ldg(X + X_index) * __ldg(X + X_index);
#else
      m_val += X[X_index];
      v_val += X[X_index] * X[X_index];
#endif
    }
    m_val = BlockReduce<T>(m_storage).Sum(m_val);
    v_val = BlockReduce<T>(v_storage).Sum(v_val);
    if (threadIdx.x == 0) {
      const T mu = m_val * scale;
      mean[i] = mu;
      variance[i] = v_val * scale - mu * mu;
    }
    __syncthreads();
  }
}

template <typename T, int D>
__global__ void MomentsCUDAKernel(
    const int outer_size,
    const int inner_size,
    SimpleArray<int, D> X_strides,
    SimpleArray<FIXED_DIVISOR, D> Y_dims,
    const T* X,
    T* mean,
    T* variance) {
  __shared__ typename BlockReduce<T>::TempStorage m_storage;
  __shared__ typename BlockReduce<T>::TempStorage v_storage;
  const T scale = T(1) / static_cast<T>(inner_size);
  for (int i = blockIdx.x; i < outer_size; i += gridDim.x) {
    T m_val = 0;
    T v_val = 0;
    for (int j = threadIdx.x; j < inner_size; j += blockDim.x) {
      int X_index = 0;
      int Y_index = i * inner_size + j;
#pragma unroll
      for (int d = D - 1; d >= 0; --d) {
        int r;
        FIXED_DIVISOR_DIV_MOD(Y_dims.data[d], Y_index, &Y_index, &r);
        X_index += r * X_strides.data[d];
      }
#if __CUDA_ARCH__ >= 350 || defined(__HIPCC__)
      m_val += __ldg(X + X_index);
      v_val += __ldg(X + X_index) * __ldg(X + X_index);
#else
      m_val += X[X_index];
      v_val += X[X_index] * X[X_index];
#endif
    }
    m_val = BlockReduce<T>(m_storage).Sum(m_val);
    v_val = BlockReduce<T>(v_storage).Sum(v_val);
    if (threadIdx.x == 0) {
      const T mu = m_val * scale;
      mean[i] = mu;
      variance[i] = v_val * scale - mu * mu;
    }
    __syncthreads();
  }
}

template <typename T, int D>
CAFFE2_CUDA_EXPORT void MomentsCUDAImpl(
    const int outer_size,
    const int inner_size,
    const int* dims,
    const int* axes,
    const T* X,
    T* mean,
    T* variance,
    HIPContext* context) {
  SimpleArray<int, D> X_strides;
  SimpleArray<FIXED_DIVISOR, D> Y_dims;
  utils::ComputeTransposedStrides(D, dims, axes, X_strides.data);
  for (int i = 0; i < D; ++i) {
    Y_dims.data[i] = FIXED_DIVISOR(dims[axes[i]]);
  }
  MomentsCUDAKernel<T, D>
      <<<std::min(outer_size, CAFFE_MAXIMUM_NUM_BLOCKS),
         CAFFE_CUDA_NUM_THREADS,
         0,
         context->hip_stream()>>>(
          outer_size, inner_size, X_strides, Y_dims, X, mean, variance);
}

template <typename T>
CAFFE2_CUDA_EXPORT void MomentsCUDA(
    const int num_dims,
    const int* dims,
    const int num_axes,
    const int* axes,
    const T* X,
    T* mean,
    T* variance,
    HIPContext* context) {
  CAFFE_ENFORCE_LE(num_axes, num_dims);
  std::vector<int> Y_dims_vector(dims, dims + num_dims);
  for (int i = 0; i < num_axes; ++i) {
    Y_dims_vector[axes[i]] = 1;
  }
  const int* X_dims = dims;
  const int* Y_dims = Y_dims_vector.data();
  const int X_size =
      std::accumulate(X_dims, X_dims + num_dims, 1, std::multiplies<int>());
  const int Y_size =
      std::accumulate(Y_dims, Y_dims + num_dims, 1, std::multiplies<int>());
  if (X_size == 0) {
    Set<T, HIPContext>(Y_size, T(0), mean, context);
    Set<T, HIPContext>(Y_size, T(0), variance, context);
    return;
  }
  if (std::equal(X_dims, X_dims + num_dims, Y_dims)) {
    hipMemcpyAsync(
        mean,
        X,
        sizeof(T) * X_size,
        hipMemcpyDeviceToDevice,
        context->hip_stream());
    Set<T, HIPContext>(Y_size, T(0), variance, context);
    return;
  }
  int rows;
  int cols;
  if (utils::IsRowwiseReduce(num_dims, X_dims, Y_dims, &rows, &cols)) {
    RowwiseMomentsCUDAKernel<T>
        <<<std::min(rows, CAFFE_MAXIMUM_NUM_BLOCKS),
           CAFFE_CUDA_NUM_THREADS,
           0,
           context->hip_stream()>>>(rows, cols, X, mean, variance);
    return;
  }
  if (utils::IsColwiseReduce(num_dims, X_dims, Y_dims, &rows, &cols)) {
    ColwiseMomentsCUDAKernel<T>
        <<<std::min(rows, CAFFE_MAXIMUM_NUM_BLOCKS),
           CAFFE_CUDA_NUM_THREADS,
           0,
           context->hip_stream()>>>(rows, cols, X, mean, variance);
    return;
  }
  std::vector<int> transpose_axes(num_dims);
  utils::ComputeTransposeAxesForReduceOp(
      num_dims, num_axes, axes, transpose_axes.data());
  const int pivot = num_dims - num_axes;
  int outer_size = 1;
  for (int i = 0; i < pivot; ++i) {
    outer_size *= dims[transpose_axes[i]];
  }
  int inner_size = 1;
  for (int i = pivot; i < num_dims; ++i) {
    inner_size *= dims[transpose_axes[i]];
  }
  DISPATCH_FUNCTION_BY_VALUE_WITH_TYPE_1(
      num_dims,
      MomentsCUDAImpl,
      T,
      outer_size,
      inner_size,
      dims,
      transpose_axes.data(),
      X,
      mean,
      variance,
      context);
}

} // namespace

#define CAFFE2_SPECIALIZED_CUDA_MOMENTS(T)                           \
  template <>                                                        \
  CAFFE2_CUDA_EXPORT void Moments<T, HIPContext>(                   \
      const int num_dims,                                            \
      const int* dims,                                               \
      const int num_axes,                                            \
      const int* axes,                                               \
      const T* X,                                                    \
      T* mean,                                                       \
      T* variance,                                                   \
      HIPContext* context) {                                        \
    MomentsCUDA<T>(                                                  \
        num_dims, dims, num_axes, axes, X, mean, variance, context); \
  }
CAFFE2_SPECIALIZED_CUDA_MOMENTS(float)
#undef CAFFE2_SPECIALIZED_CUDA_MOMENTS

namespace {

template <typename T>
__global__ void
InvStdCUDAKernel(const int N, const T epsilon, const T* var, T* inv_std);

#define DELEGATE_INV_STD_KERNEL_FUNCTION(T, Func)               \
  template <>                                                   \
  __global__ void InvStdCUDAKernel<T>(                          \
      const int N, const T epsilon, const T* var, T* inv_std) { \
    CUDA_1D_KERNEL_LOOP(i, N) {                                 \
      inv_std[i] = Func(var[i] + epsilon);                      \
    }                                                           \
  }
DELEGATE_INV_STD_KERNEL_FUNCTION(float, rsqrtf)
#undef DELEGATE_INV_STD_KERNEL_FUNCTION

} // namespace

#define CAFFE2_SPECIALIZED_CUDA_INV_STD(T)                      \
  template <>                                                   \
  CAFFE2_CUDA_EXPORT void InvStd<T, HIPContext>(               \
      const int N,                                              \
      const T epsilon,                                          \
      const T* var,                                             \
      T* inv_std,                                               \
      HIPContext* context) {                                   \
    InvStdCUDAKernel<T>                                         \
        <<<CAFFE_GET_BLOCKS(N),                                 \
           CAFFE_CUDA_NUM_THREADS,                              \
           0,                                                   \
           context->hip_stream()>>>(N, epsilon, var, inv_std); \
  }
CAFFE2_SPECIALIZED_CUDA_INV_STD(float)
#undef CAFFE2_SPECIALIZED_CUDA_INV_STD

namespace {

constexpr int kTileDim = 32;
constexpr int kBlockRows = 8;

// Splits the original matrix into submatrices with size 32 * 32.
// Each block transposes one submatrix by loading it into shared memory.
// Reference https://devblogs.nvidia.com/efficient-matrix-transpose-cuda-cc/
template <typename T>
__global__ void BatchTranspose2DCUDAKernel(
    const int N,
    const int H,
    const int W,
    const T* X,
    T* Y) {
  __shared__ T tile[kTileDim][kTileDim + 1];
  const int h = (H + kTileDim - 1) / kTileDim;
  const int w = (W + kTileDim - 1) / kTileDim;
  const int outer_size = N * h * w;
  for (int i = blockIdx.x; i < outer_size; i += gridDim.x) {
    const int n = i / (h * w);
    const int k = i % (h * w);
    const int r = k / w;
    const int c = k % w;
    const int offset = n * H * W;
    int x = c * kTileDim + threadIdx.x;
    int y = r * kTileDim + threadIdx.y;
    if (x < W) {
      for (int j = 0; j < kTileDim && y + j < H; j += kBlockRows) {
#if __CUDA_ARCH__ >= 350 || defined(__HIPCC__)
        tile[threadIdx.y + j][threadIdx.x] =
            __ldg(X + offset + (y + j) * W + x);
#else
        tile[threadIdx.y + j][threadIdx.x] = X[offset + (y + j) * W + x];
#endif
      }
    }
    __syncthreads();
    x = r * kTileDim + threadIdx.x;
    y = c * kTileDim + threadIdx.y;
    if (x < H) {
      for (int j = 0; j < kTileDim && y + j < W; j += kBlockRows) {
        Y[offset + (y + j) * H + x] = tile[threadIdx.x][threadIdx.y + j];
      }
    }
    __syncthreads();
  }
}

template <typename T, int D>
__global__ void TransposeCUDAKernel(
    const int size,
    const SimpleArray<int, D> X_strides,
    const SimpleArray<FIXED_DIVISOR, D> Y_dims,
    const T* X,
    T* Y) {
  CUDA_1D_KERNEL_LOOP(Y_index, size) {
    int X_index = 0;
    int Y_index_val = Y_index;
#pragma unroll
    for (int i = D - 1; i >= 0; --i) {
      int d;
      FIXED_DIVISOR_DIV_MOD(Y_dims.data[i], Y_index_val, &Y_index_val, &d);
      X_index += d * X_strides.data[i];
    }
#if __CUDA_ARCH__ >= 350 || defined(__HIPCC__)
    Y[Y_index] = __ldg(X + X_index);
#else
    Y[Y_index] = X[X_index];
#endif
  }
}

template <typename T, int D>
CAFFE2_CUDA_EXPORT void TransposeCUDAImpl(
    const int* dims,
    const int* axes,
    const T* X,
    T* Y,
    HIPContext* context) {
  SimpleArray<int, D> X_strides;
  SimpleArray<FIXED_DIVISOR, D> Y_dims;
  utils::ComputeTransposedStrides(D, dims, axes, X_strides.data);
  int size = 1;
  for (int i = 0; i < D; ++i) {
    Y_dims.data[i] = FIXED_DIVISOR(dims[axes[i]]);
    size *= dims[i];
  }
  TransposeCUDAKernel<T, D>
      <<<CAFFE_GET_BLOCKS(size),
         CAFFE_CUDA_NUM_THREADS,
         0,
         context->hip_stream()>>>(size, X_strides, Y_dims, X, Y);
}

} // namespace

#define CAFFE2_SPECIALIZED_CUDA_TRANSPOSE(T)                                 \
  template <>                                                                \
  CAFFE2_CUDA_EXPORT void Transpose<T, HIPContext>(                         \
      const int ndim,                                                        \
      const int* dims,                                                       \
      const int* axes,                                                       \
      const T* X,                                                            \
      T* Y,                                                                  \
      HIPContext* context) {                                                \
    if (utils::IsIdentityPermutation(ndim, axes)) {                          \
      const int size =                                                       \
          std::accumulate(dims, dims + ndim, 1, std::multiplies<int>());     \
      context->template CopySameDevice<T>(size, X, Y);                       \
      return;                                                                \
    }                                                                        \
    if (utils::IsBatchTranspose2D(ndim, axes)) {                             \
      const int N =                                                          \
          std::accumulate(dims, dims + ndim - 2, 1, std::multiplies<int>()); \
      const int H = dims[ndim - 2];                                          \
      const int W = dims[ndim - 1];                                          \
      const int h = (H + kTileDim - 1) / kTileDim;                           \
      const int w = (W + kTileDim - 1) / kTileDim;                           \
      const int outer_size = N * h * w;                                      \
      const dim3 dim_block(kTileDim, kBlockRows, 1);                         \
      BatchTranspose2DCUDAKernel<T>                                          \
          <<<std::min(outer_size, CAFFE_MAXIMUM_NUM_BLOCKS),                 \
             dim_block,                                                      \
             0,                                                              \
             context->hip_stream()>>>(N, H, W, X, Y);                       \
      return;                                                                \
    }                                                                        \
    DISPATCH_FUNCTION_BY_VALUE_WITH_TYPE_1(                                  \
        ndim, TransposeCUDAImpl, T, dims, axes, X, Y, context);              \
  }
CAFFE2_SPECIALIZED_CUDA_TRANSPOSE(float)
CAFFE2_SPECIALIZED_CUDA_TRANSPOSE(double)
CAFFE2_SPECIALIZED_CUDA_TRANSPOSE(int)
CAFFE2_SPECIALIZED_CUDA_TRANSPOSE(int64_t)
#undef CAFFE2_SPECIALIZED_CUDA_TRANSPOSE

namespace {

template <typename T, StorageOrder kOrder>
__global__ void AffineChannelCUDAKernel(
    const int size,
    const int C,
    const int HxW,
    const T* X,
    const T* scale,
    const T* bias,
    T* Y) {
  CUDA_1D_KERNEL_LOOP(i, size) {
    const int c = kOrder == StorageOrder::NCHW ? i / HxW % C : i % C;
#if __CUDA_ARCH__ >= 350 || defined(__HIPCC__)
    Y[i] = __ldg(scale + c) * __ldg(X + i) + __ldg(bias + c);
#else
    Y[i] = scale[c] * X[i] + bias[c];
#endif
  }
}

} // namespace

#define CAFFE2_SPECIALIZED_CUDA_AFFINE_CHANNEL(T, kOrder)              \
  template <>                                                          \
  CAFFE2_CUDA_EXPORT void AffineChannel<T, HIPContext, kOrder>(       \
      const int N,                                                     \
      const int C,                                                     \
      const int HxW,                                                   \
      const T* X,                                                      \
      const T* scale,                                                  \
      const T* bias,                                                   \
      T* Y,                                                            \
      HIPContext* context) {                                          \
    const int size = N * C * HxW;                                      \
    AffineChannelCUDAKernel<T, kOrder>                                 \
        <<<CAFFE_GET_BLOCKS(size),                                     \
           CAFFE_CUDA_NUM_THREADS,                                     \
           0,                                                          \
           context->hip_stream()>>>(size, C, HxW, X, scale, bias, Y); \
  }
CAFFE2_SPECIALIZED_CUDA_AFFINE_CHANNEL(float, StorageOrder::NCHW)
CAFFE2_SPECIALIZED_CUDA_AFFINE_CHANNEL(float, StorageOrder::NHWC)
#undef CAFFE2_SPECIALIZED_CUDA_AFFINE_CHANNEL

#define CAFFE2_SPECIALIZED_CUDA_NCHW2NHWC(T)               \
  template <>                                              \
  CAFFE2_CUDA_EXPORT void NCHW2NHWC<T, HIPContext>(       \
      const int N,                                         \
      const int C,                                         \
      const int HxW,                                       \
      const T* X,                                          \
      T* Y,                                                \
      HIPContext* context) {                              \
    const int h = (C + kTileDim - 1) / kTileDim;           \
    const int w = (HxW + kTileDim - 1) / kTileDim;         \
    const int outer_size = N * h * w;                      \
    const dim3 dim_block(kTileDim, kBlockRows, 1);         \
    BatchTranspose2DCUDAKernel<T>                          \
        <<<std::min(outer_size, CAFFE_MAXIMUM_NUM_BLOCKS), \
           dim_block,                                      \
           0,                                              \
           context->hip_stream()>>>(N, C, HxW, X, Y);     \
  }
CAFFE2_SPECIALIZED_CUDA_NCHW2NHWC(float)
#undef CAFFE2_SPECIALIZED_CUDA_NCHW2NHWC

#define CAFFE2_SPECIALIZED_CUDA_NHWC2NCHW(T)               \
  template <>                                              \
  CAFFE2_CUDA_EXPORT void NHWC2NCHW<T, HIPContext>(       \
      const int N,                                         \
      const int C,                                         \
      const int HxW,                                       \
      const T* X,                                          \
      T* Y,                                                \
      HIPContext* context) {                              \
    const int h = (HxW + kTileDim - 1) / kTileDim;         \
    const int w = (C + kTileDim - 1) / kTileDim;           \
    const int outer_size = N * h * w;                      \
    const dim3 dim_block(kTileDim, kBlockRows, 1);         \
    BatchTranspose2DCUDAKernel<T>                          \
        <<<std::min(outer_size, CAFFE_MAXIMUM_NUM_BLOCKS), \
           dim_block,                                      \
           0,                                              \
           context->hip_stream()>>>(N, HxW, C, X, Y);     \
  }
CAFFE2_SPECIALIZED_CUDA_NHWC2NCHW(float)
#undef CAFFE2_SPECIALIZED_CUDA_NHWC2NCHW

} // namespace math
} // namespace caffe2
